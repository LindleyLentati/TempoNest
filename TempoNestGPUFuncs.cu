#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "/usr/include/gsl/gsl_sf_gamma.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cula_lapack_device.h>
#include <cula_blas_device.h>


#define BLOCK_SIZE 16

double *GlobalGmat_d;
float *GlobalGmatFloat_d;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)

double iter_factorialGPU(unsigned int n)
{
    double ret = 1;
    for(unsigned int i = 1; i <= n; ++i)
        ret *= i;
    return ret;
}


void checkStatus(culaStatus status)
{
    char buf[256];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    culaShutdown();
    exit(EXIT_FAILURE);
}

void checkStatusCarryOn(culaStatus status, int &CarryOn)
{
    char buf[256];

    if(!status){
    	CarryOn=0;
        return;
        }

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    //printf("%s\n", buf);
    CarryOn=1;
    
    culaShutdown();
    culaStatus status2;
	status2 = culaInitialize();
	
}


void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    culaShutdown();
    exit(EXIT_FAILURE);
}


// simple kernel function that adds two vectors
__global__ void vect_add(double *a, double *b, int N)
{
   int Bidx = blockIdx.x;
   //a[Bidx*N+Tidx] = a[Bidx*N+Tidx] + b[Bidx*N+Tidx]; 
   for(int i =0; i < N; i++){
   		a[Bidx*N+i] = a[Bidx*N+i] + b[Bidx*N+i]; 
   		}
}


// simple kernel function that calcs det of a matrix
__global__ void calc_det(double *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void Floatcalc_det(float *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void calc_detFloat(float *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

void CPUMakecov(double *A_d, double *BATvec, double *NoiseVec, double *SpecParm, int Aheight, int Awidth, double *FactorialList) {

	// Each thread computes one element of C
	// by accumulating results into Cvalue
	

	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecParm[1];
	double gwampsquared=SpecParm[0];
	double covconst=SpecParm[2];
	double CovVal=0;

	for(int row=0;row<Aheight; row++){
		for(int col=0;col<Awidth; col++){
			double timdiff= BATvec[row] - BATvec[col];	
			double tau=2.0*M_PI*fabs(timdiff);
			double covsum=0;

			for(int k=0; k <=10; k++){
		
					covsum=covsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(FactorialList[2*k]*(2*k+1-modelalpha));

			}

			//A_d[row * Awidth + col]=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);
			CovVal=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);

			if(row==col){
				//A_d[row * Awidth + col] += NoiseVec[row];
				CovVal += NoiseVec[row];
				
			}
				//printf("Make: %i %i %g %i %g  \n", row,col,CovVal,row+col, A_d[row * Awidth + col]);
		}
	}
}

__global__ void Makecov(double *A_d, double *BATvec, double *NoiseVec, double *SpecParm, int Aheight, int Awidth) {

	// Each thread computes one element of C
	// by accumulating results into Cvalue
	

	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecParm[1];
	double gwampsquared=SpecParm[0];
	double covconst=SpecParm[2];
	

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;
	double timdiff= BATvec[row] - BATvec[col];	
	double tau=2.0*M_PI*fabs(timdiff);
	double covsum=0;

	for(int k=0; k <=5; k++){
	
		    double ret = 1;
			for(unsigned int i = 1; i <= 2*k; ++i){
				ret *= (double)i;
			}
    
			covsum=covsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(ret*(2*k+1-modelalpha));

	}

	A_d[row * Awidth + col]=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);

	if(row==col){
		A_d[row * Awidth + col] += NoiseVec[row];
	}

}

__global__ void MatMulKernel(int Arow,int Acol,int Brow, int Bcol,double *A,double *B,double *C)
{

	int Crow=Arow;
	int Ccol=Bcol;
    double Ctemp = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    __syncthreads();

	if(row < Arow && col < Bcol) {

   		Ctemp = A[row] * B[col * Brow + row];
						  //GGTest[col*N + row]


	   C[col*Crow+row] = Ctemp;
	}
	  // 
}

__global__ void NoiseMulKernel(double *A_d,double *B_d,double *C_d, int Aheight, int Awidth, int Bheight, int Bwidth) {

	// Each thread computes one element of C
	// by accumulating results into Cvalue
	
	int Cwidth=Bwidth;
	int Cheight=Aheight;
	double Cvalue = 0.0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > Aheight || col > Bwidth) return;

	Cvalue = (A_d[row]) * (B_d[row * Bwidth + col]);		
	C_d[row * Cwidth + col] = Cvalue;
}


// simple kernel function that does diag mult for noise
__global__ void mult_noise(double *Mat_d,double *Noise_d,double *NM_d,int N, int M)
{

	int Bidx = blockIdx.x;
	for(int i=0;i<M; i++){NM_d[i*N + Bidx]=Mat_d[i*N + Bidx]*Noise_d[Bidx];}
   		
}


extern "C" void WhiteMarginGPUWrapper_(double *Noise, double *Res, double *likeInfo, int N, int G)
{

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	
	double *GRes;
	GRes = (double*)malloc(sizeof(double)*G);
	double *WorkingGRes;
	WorkingGRes = (double*)malloc(sizeof(double)*G);

	// declare GPU copies
	double *Res_d;
	double *Noise_d;
	double *dettemp_d;
	
	double *NG_d;
	double *GG_d;
	double *GRes_d;



    hipError_t err;
    culaStatus status;

   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
 	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
	 
 	 err = hipMalloc( (void **)&NG_d, sizeof(double)*N*G);
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&GG_d, sizeof(double)*G*G);
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&GRes_d, sizeof(double)*G*G);
	 checkCudaError(err);


   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	 err = hipMemcpy(Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x,	(N + dimBlock.y - 1) / dimBlock.y);
	NoiseMulKernel<<<dimGrid, dimBlock>>>(Noise_d,GlobalGmat_d,NG_d,N,N,N,G);
 	
 //	mult_noise<<< N, 1 >>>(GlobalGmat_d,Noise_d,NG_d,N,G);
 	
 	double alpha=1.0;
 	double beta=0.0; 
 	status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, NG_d, N, beta, GG_d, G);
	checkStatus(status);


 	 status = culaDeviceDpotrf('L', G, GG_d, G);
     checkStatus(status);

     calc_det<<< 1, 1 >>>( GG_d, dettemp_d, G);
     err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     likeInfo[0]=dettemp[0];
	
	 status = culaDeviceDgemv('T', N, G, alpha, GlobalGmat_d, N, Res_d, 1, beta, GRes_d, 1);
     checkStatus(status);
 	 err = hipMemcpy(GRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

 	 status=culaDeviceDpotrs('L', G, 1, GG_d, G, GRes_d, G);
	 checkStatus(status);
	 err = hipMemcpy(WorkingGRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
	 
	 double sum=0;
	 for(int i=0; i<G;i++){sum=sum+GRes[i]*WorkingGRes[i];}
	 likeInfo[1]=sum;
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
	 hipFree(Noise_d);
	 hipFree(NG_d);
	 hipFree(GG_d);
	 hipFree(GRes_d);

	 
  	 free(GRes); 
  	 free(dettemp);
  	 free(WorkingGRes);


   return;
}


extern "C" void vHRedMarginGPUWrapper2_(double *Res, double *BatVec, double *NoiseVec, double *SpecInfo, double *likeInfo, double *FactorialList, int N, int G)
{


    hipError_t err;
    culaStatus status;
    
    double secday=24*60*60;
	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecInfo[1];
	double covconst=gsl_sf_gamma(1-modelalpha)*sin(0.5*M_PI*modelalpha);
	
	double gwamp=pow(10.0,SpecInfo[0]);
	double gwampsquared=gwamp*gwamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-modelalpha)))/(pow(flo,(modelalpha-1)));
	
	SpecInfo[0]=gwampsquared;
	SpecInfo[2]=covconst;
	
	double *Res_d;
	double *BatVec_d;
	double *NoiseVec_d;
	double *SpecInfo_d;
	double *CovMatrix_d;

  
  	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&BatVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&NoiseVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&SpecInfo_d, sizeof(double)*3 );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 

     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( BatVec_d, BatVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( NoiseVec_d, NoiseVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy(SpecInfo_d, SpecInfo, sizeof(double)*3, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;//((G + dimBlock.x - 1) / dimBlock.x,(N + dimBlock.y - 1) / dimBlock.y);
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
	 Makecov<<<dimGrid, dimBlock>>>(CovMatrix_d, BatVec_d, NoiseVec_d, SpecInfo_d, N,N);
	 
	// 	double *CovMatrix;
//	CovMatrix = (double*)malloc(sizeof(double)*N*N);
 //	err = hipMemcpy( CovMatrix, CovMatrix_d, sizeof(double)*N*N, hipMemcpyDeviceToHost );
 	//checkCudaError(err);
	  

	//CPUMakecov(CovMatrix, BatVec, NoiseVec, SpecInfo, N,N, FactorialList);
	//err = hipMemcpy( CovMatrix_d, CovMatrix, sizeof(double)*N*N, hipMemcpyHostToDevice );
 	//checkCudaError(err);
 	//free(CovMatrix);
 	 
	//double *CovMatrix;
	//CovMatrix = (double*)malloc(sizeof(double)*N*N);	
  //   err = hipMemcpy( CovMatrix, CovMatrix_d, sizeof(double)*N*N, hipMemcpyDeviceToHost );
 //	 checkCudaError(err);
//	printf("%g \n",CovMatrix[0]);
	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	
	double *GRes;
	GRes =(double*)malloc(sizeof(double)*G);
	double *WorkingGRes;
	WorkingGRes =(double*)malloc(sizeof(double)*G);
	

	// declare GPU copies

	double *CG_d;
	double *GCG_d;
	double *GRes_d;
	double *dettemp_d;


   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);

	 
	  	 err = hipMalloc( (void **)&CG_d, sizeof(double)*N*G );
	 checkCudaError(err);
	  	 err = hipMalloc( (void **)&GCG_d, sizeof(double)*G*G );
	 checkCudaError(err);
	  	 err = hipMalloc( (void **)&GRes_d, sizeof(double)*G );
	 checkCudaError(err);

   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	// err = hipMemcpy(CovMatrix_d, CovMatrix, sizeof(double)*N*N, hipMemcpyHostToDevice );
 	// checkCudaError(err);

 	 
	int carryOn=0;
  	double alpha=1.0;
 	double beta=0.0; 

 	status =  culaDeviceDsymm('L', 'U', N, G, alpha, CovMatrix_d, N, GlobalGmat_d, N, beta, CG_d, N);
	checkStatus(status);
	//printf("done first linalg\n");
  	status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, CG_d, N, beta, GCG_d, G);
	checkStatus(status);

	 status = culaDeviceDgemv('T', N, G, alpha, GlobalGmat_d, N, Res_d, 1, beta, GRes_d, 1);
     checkStatus(status);
 	 err = hipMemcpy(GRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);	 
 	 //	printf("do chol\n");
		
 	 status = culaDeviceDpotrf('L', G, GCG_d, G);
     checkStatusCarryOn(status,carryOn);

     calc_det<<< 1, 1 >>>( GCG_d, dettemp_d, G);
     err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     likeInfo[0]=dettemp[0];
    // printf("det: %g \n",likeInfo[0]);
    
              if(carryOn == 1){
     	// printf("Bad chol\n");
     	 
     	 	 hipFree(dettemp_d);
 	 hipFree(Res_d);
 	 hipFree(BatVec_d);
 	 hipFree(NoiseVec_d);
 	 hipFree(SpecInfo_d);
	 hipFree(CovMatrix_d);
 	 hipFree(CG_d);
	 hipFree(GCG_d);
	 hipFree(GRes_d);

  	 free(dettemp);
   	 free(GRes);
  	 free(WorkingGRes);
  	 
  	 
  	 return;

  	 }

 	 status=culaDeviceDpotrs('L', G, 1, GCG_d, G, GRes_d, G);
	 checkStatus(status);
	 err = hipMemcpy(WorkingGRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

	 double sum=0;
	 for(int i=0; i<G;i++){sum=sum+GRes[i]*WorkingGRes[i];}
	 likeInfo[1]=sum;
	 //printf("like: %g \n",sum);
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
  	 hipFree(BatVec_d);
 	 hipFree(NoiseVec_d);
 	 hipFree(SpecInfo_d);
	 hipFree(CovMatrix_d);
 	 hipFree(CG_d);
	 hipFree(GCG_d);
	 hipFree(GRes_d);

  	 free(dettemp);
   	 free(GRes);
  	 free(WorkingGRes);
  


   return;
}

extern "C" void vHRedGPUWrapper_(double *SpecInfo, double *BatVec, double *Res, double *NoiseVec, double *likeInfo, int N)
{

	double secday=24*60*60;
	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecInfo[1];
	double gwamp=pow(10.0,SpecInfo[0]);
	double gwampsquared=gwamp*gwamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-modelalpha)))/(pow(flo,(modelalpha-1)));

	double timdiff=0;

	double covconst=gsl_sf_gamma(1-modelalpha)*sin(0.5*M_PI*modelalpha);
// 	printf("constants: %g %g \n",gwampsquared,covconst);


	
	double *CovMatrix = new double[N*N];

	for(int o1=0;o1<N; o1++){

		for(int o2=0;o2<N; o2++){
			timdiff=BatVec[o1]-BatVec[o2];	
			double tau=2.0*M_PI*fabs(timdiff);
			double covsum=0;

			for(int k=0; k <=4; k++){
				
				covsum=covsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(iter_factorialGPU(2*k)*(2*k+1-modelalpha));
				//printf("Fac: %i %.10g \n", k, covsum);

			}

			CovMatrix[o1*N + o2]=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);
// 			printf("%i %i %g %g %g\n",o1,o2,CovMatrix[o1][o2],fabs(timdiff),covsum);

			if(o1==o2){
				CovMatrix[o1*N + o2] += NoiseVec[o1];
			}

		}
	}
	

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	

	double *WorkingRes;
	WorkingRes = (double*)malloc(sizeof(double)*N);

	// declare GPU copies
	double *Res_d;
	double *CovMatrix_d;
	double *dettemp_d;

    hipError_t err;
    culaStatus status;

   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
 	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);

   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	 err = hipMemcpy(CovMatrix_d, CovMatrix, sizeof(double)*N*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

		int carryOn=0;
 	 status = culaDeviceDpotrf('L', N, CovMatrix_d, N);
     checkStatusCarryOn(status,carryOn);
     


     calc_det<<< 1, 1 >>>( CovMatrix_d, dettemp_d, N);
     err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     likeInfo[0]=dettemp[0];
     
          if(carryOn == 1){
     	// printf("Bad chol\n");
     	 hipFree(dettemp_d);
	 	 hipFree(Res_d);
		 hipFree(CovMatrix_d);
		delete[] CovMatrix;
	  	 free(dettemp);
	  	 free(WorkingRes);
	  	 return;
  	 }
	
	//status = culaDeviceDpotri('L', N, CovMatrix_d, N);
 	 status=culaDeviceDpotrs('L', N, 1, CovMatrix_d, N, Res_d, N);
	 checkStatus(status);
	 err = hipMemcpy(WorkingRes, Res_d, sizeof(double)*N, hipMemcpyDeviceToHost);
  	 //err = hipMemcpy(CovMatrix, CovMatrix_d, sizeof(double)*N*N, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
//	                for(int i=0; i<N;i++){
 //                
 //                       for(int j=0;j<i;j++){
 //                              CovMatrix[j*N +i]=CovMatrix[i*N + j];
 //                        }
 //               }
	 double sum=0;
//	 for(int i=0; i<N;i++){
//	 	 for(int j=0; j<N;j++){
//	 		sum=sum+Res[i]*CovMatrix[i*N + j]*Res[j];
////	 		}
//	 	}
	 	
	 	for(int i=0; i<N;i++){
	 		sum=sum+Res[i]*WorkingRes[i];
	 		}
	 likeInfo[1]=sum;
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
	 hipFree(CovMatrix_d);

  	 free(dettemp);
  	 free(WorkingRes);
  	 
  	 delete[] CovMatrix;


   return;
}


extern "C" void vHRedMarginGPUWrapper_(double *CovMatrix, double *Res, double *likeInfo, int N, int G)
{

	//printf("Entered \n");
	clock_t startClock,endClock;
	startClock = clock();
	
	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	

	double *WorkingGRes;
	WorkingGRes = (double*)malloc(sizeof(double)*G);
	double *GRes;
	GRes = (double*)malloc(sizeof(double)*G);
	
	// declare GPU copies
	double *Res_d;
	double *CovMatrix_d;
	double *dettemp_d;
	double *CG_d;
	double *GCG_d;
	double *GRes_d;

    hipError_t err;
    culaStatus status;

   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
 	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CG_d, sizeof(double)*N*G );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&GCG_d, sizeof(double)*G*G);
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GRes_d, sizeof(double)*G);
	 checkCudaError(err);
	 
	// printf("Copying \n");
   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	 err = hipMemcpy(CovMatrix_d, CovMatrix, sizeof(double)*N*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 
  //	endClock = clock();
   //	printf("Copied: time taken = %.2f (s)\n",(endClock-startClock)/(double)CLOCKS_PER_SEC);
 	// startClock = clock();

  	double alpha=1.0;
 	double beta=0.0; 
 	
 	
 	status =  culaDeviceDgemm('N', 'N', N, G, N, alpha, CovMatrix_d, N, GlobalGmat_d, N, beta, CG_d, N);
	checkStatus(status);

  	status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, CG_d, N, beta, GCG_d, G);
	checkStatus(status);

	 status = culaDeviceDgemv('T', N, G, alpha, GlobalGmat_d, N, Res_d, 1, beta, GRes_d, 1);
     checkStatus(status);
 	 err = hipMemcpy(GRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);	 
 	 
	int carryOn=0;
 	 status = culaDeviceDpotrf('L', G, GCG_d, G);
     checkStatusCarryOn(status,carryOn);


     calc_det<<< 1, 1 >>>( GCG_d, dettemp_d, G);
     err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     likeInfo[0]=dettemp[0];
	 
	      if(carryOn==1){
     	//printf("bad chol \n");
 		 hipFree(dettemp_d);
	 	 hipFree(Res_d);
		 hipFree(CovMatrix_d);
	 	 hipFree(CG_d);
		 hipFree(GCG_d);
		 hipFree(GRes_d);

	  	 free(dettemp);
	  	 free(GRes);
	  	 free(WorkingGRes);
  	 	return;
  	 	}
  	 	
  	 	
 	 status=culaDeviceDpotrs('L', G, 1, GCG_d, G, GRes_d, G);
	 checkStatus(status);
	 err = hipMemcpy(WorkingGRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

	 double sum=0;
	 for(int i=0; i<G;i++){sum=sum+GRes[i]*WorkingGRes[i];}
	 likeInfo[1]=sum;
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
	 hipFree(CovMatrix_d);
 	 hipFree(CG_d);
	 hipFree(GCG_d);
	 hipFree(GRes_d);

  	 free(dettemp);
  	 free(GRes);
  	 free(WorkingGRes);
  	 
  	 	//printf("Done: time taken = %.2f (s)\n",(endClock-startClock)/(double)CLOCKS_PER_SEC);


   return;
}

// simple kernel function that calculates the FMatrix
__global__ void make_fmatrix(double *FMatrix_d,double *Freqs_d, double *BATvec_d, int N,int F)
{

	int Bidx = blockIdx.x;
	
	for(int i=0;i<F/2;i++){
		
			FMatrix_d[i*N + Bidx]=cos(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
			FMatrix_d[(i+F/2)*N + Bidx]=sin(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
	}


}



extern "C" void LRedGPUWrapper_(double *Freqs, double *resvec, double *BATvec, double *Noise, double **FNF, double *NFd, int N, int F){

	double *Freqs_d;
	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	
	double *FMatrix_d;
	double *NF_d;	
	double *FNF_d;
	double *NFd_d;
	
	double *FNFvec;
	FNFvec = (double*)malloc(sizeof(double)*F*F);

    hipError_t err;
    culaStatus status;
      
  	 err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
	 
   	 err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&NF_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&FNF_d, sizeof(double)*F*F );
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&NFd_d, sizeof(double)*F);
	 checkCudaError(err);	
	 
   // copy vectors from CPU to GPU
   	 err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,N,F);

 	 mult_noise<<< N, 1 >>>(FMatrix_d,Noise_d,NF_d,N,F);
 	 

 	 
  	double alpha=1.0;
 	double beta=0.0; 
 	status =  culaDeviceDgemm('T', 'N', F, F, N, alpha, FMatrix_d, N, NF_d, N, beta, FNF_d, F);
	checkStatus(status);
	
 	 status = culaDeviceDgemv('T', N, F, alpha, NF_d, N, resvec_d, 1, beta, NFd_d, 1);
     checkStatus(status);
	
	 err = hipMemcpy(FNFvec, FNF_d, sizeof(double)*F*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 	
	 err = hipMemcpy(NFd, NFd_d, sizeof(double)*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

		for(int f1=0;f1<F; f1++){
			for(int f2=0;f2<F; f2++){
 
				FNF[f2][f1]=FNFvec[f1*F + f2];
			}
		}

	hipFree(Freqs_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
    hipFree(FMatrix_d);
    hipFree(NF_d);
	hipFree(FNF_d);
	hipFree(resvec_d);
	hipFree(NFd_d);
	
	free(FNFvec);
		
}
 	 


extern "C" void LRedMarginGPUWrapper_(double *Freqs, double *resvec, double *BATvec, double *Noise, double **FNF, double *NFd, double *likeVals, int N, int F, int G){

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	double *dettemp_d;

	double *Freqs_d;
	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	
		
	double *NG_d;
	double *GG_d;
	double *GNG_d;
	double *GNGd_d;
	
	double *GNGd;
	GNGd = (double*)malloc(sizeof(double)*N);

	
	double *FMatrix_d;
	double *NF_d;	
	double *FNF_d;
	double *NFd_d;
	
	double *FNFvec;
	FNFvec = (double*)malloc(sizeof(double)*F*F);

    hipError_t err;
    culaStatus status;
    
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
      
  	 err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GNGd_d, sizeof(double)*N );
	 checkCudaError(err);

	 
   	 err = hipMalloc( (void **)&NG_d, sizeof(double)*N*G);
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GG_d, sizeof(double)*G*G );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&GNG_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 
   	 err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&NF_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&FNF_d, sizeof(double)*F*F );
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&NFd_d, sizeof(double)*F);
	 checkCudaError(err);	
	 
   // copy vectors from CPU to GPU
   	 err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x,(N + dimBlock.y - 1) / dimBlock.y);
	 NoiseMulKernel<<<dimGrid, dimBlock>>>(Noise_d,GlobalGmat_d,NG_d,N,N,N,G);
	 

 	 double alpha=1.0;
 	 double beta=0.0; 
 	 status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, NG_d, N, beta, GG_d, G);
	 checkStatus(status);


 	 status = culaDeviceDpotrf('L', G, GG_d, G);
     checkStatus(status);

     calc_det<<< 1, 1 >>>( GG_d, dettemp_d, G);
     err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     likeVals[0]=dettemp[0];
     
  	 status = culaDeviceDpotri('L', G, GG_d, G);
     checkStatus(status);
     
  	 status =  culaDeviceDsymm('R', 'L', N, G, alpha, GG_d, G, GlobalGmat_d, N, beta, NG_d, N);
	 checkStatus(status);
	 
  	 status =  culaDeviceDgemm('N', 'T', N, N, G, alpha, NG_d, N, GlobalGmat_d, N, beta, GNG_d, N);
	 checkStatus(status);
	 
  	 status = culaDeviceDgemv('N', N, N, alpha, GNG_d, N, resvec_d, 1, beta, GNGd_d, 1);
     checkStatus(status);
     

	 err = hipMemcpy(GNGd, GNGd_d, sizeof(double)*N, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 likeVals[1]=0;
  	 for(int i =0;i < N; i++){likeVals[1] += resvec[i]*GNGd[i];}

 	 make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,N,F);
 	 
   	 status =  culaDeviceDgemm('N', 'N', N, F, N, alpha, GNG_d, N, FMatrix_d, N, beta, NF_d, N);
	 checkStatus(status);
	 
  	 status =  culaDeviceDgemm('T', 'N', F, F, N, alpha, FMatrix_d, N, NF_d, N, beta, FNF_d, F);
	 checkStatus(status);
	
 	 status = culaDeviceDgemv('T', N, F, alpha, NF_d, N, resvec_d, 1, beta, NFd_d, 1);
     checkStatus(status);
	
	 err = hipMemcpy(FNFvec, FNF_d, sizeof(double)*F*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 	
	 err = hipMemcpy(NFd, NFd_d, sizeof(double)*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

		for(int f1=0;f1<F; f1++){
			for(int f2=0;f2<F; f2++){
 
				FNF[f2][f1]=FNFvec[f1*F + f2];
				//printf("GPUFNF: %i %i %g \n",f1,f2);
			}
		}

	hipFree(dettemp_d);
	hipFree(Freqs_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
    hipFree(FMatrix_d);
    hipFree(NF_d);
	hipFree(FNF_d);
	hipFree(resvec_d);
	hipFree(NFd_d);
	
    hipFree(NG_d);
    hipFree(GG_d);
	hipFree(GNG_d);
	hipFree(GNGd_d);

	
	free(FNFvec);
	free(dettemp);
	free(GNGd);
	
}
 	 

 extern "C" void copy_floatgmat_(float *G, int N){

    hipError_t err;

   // Allocate memory on GPU
	printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalGmatFloat_d, sizeof(float)*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalGmatFloat_d, G, sizeof(float)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}


extern "C" void copy_gmat_(double *G, int N){

    hipError_t err;

   // Allocate memory on GPU
	printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalGmat_d, sizeof(double)*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalGmat_d, G, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}

