#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "/usr/include/gsl/gsl_sf_gamma.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cula_lapack_device.h>
#include <cula_blas_device.h>


#define BLOCK_SIZE 16

double *GlobalGmat_d;
float *GlobalGmatFloat_d;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)

double iter_factorialGPU(unsigned int n)
{
    double ret = 1;
    for(unsigned int i = 1; i <= n; ++i)
        ret *= i;
    return ret;
}


void checkStatus(culaStatus status)
{
    char buf[256];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    culaShutdown();
    exit(EXIT_FAILURE);
}

void checkStatusCarryOn(culaStatus status, int &CarryOn)
{
    char buf[256];

    if(!status){
    	CarryOn=0;
        return;
        }

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    //printf("%s\n", buf);
    CarryOn=1;
    
    culaShutdown();
    culaStatus status2;
	status2 = culaInitialize();
	
}


void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    culaShutdown();
    exit(EXIT_FAILURE);
}


// simple kernel function that adds two vectors
__global__ void vect_add(double *a, double *b, int N)
{
   int Bidx = blockIdx.x;
   //a[Bidx*N+Tidx] = a[Bidx*N+Tidx] + b[Bidx*N+Tidx]; 
   for(int i =0; i < N; i++){
   		a[Bidx*N+i] = a[Bidx*N+i] + b[Bidx*N+i]; 
   		}
}


// simple kernel function that calcs det of a matrix
__global__ void calc_det(double *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void Floatcalc_det(float *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void calc_detFloat(float *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

__global__ void Makecov(double *A_d, double *BATvec, double *NoiseVec, double *SpecParm, int Aheight, int Awidth) {

	// Each thread computes one element of C
	// by accumulating results into Cvalue
	

	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecParm[1];
	double gwampsquared=SpecParm[0];
	double covconst=SpecParm[2];
	

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;
	double timdiff= BATvec[row] - BATvec[col];	
	double tau=2.0*M_PI*fabs(timdiff);
	double covsum=0;

	for(int k=0; k <=5; k++){
	
		    double ret = 1;
			for(unsigned int i = 1; i <= 2*k; ++i){
				ret *= (double)i;
			}
    
			covsum=covsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(ret*(2*k+1-modelalpha));

	}

	A_d[row * Awidth + col]=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);

	if(row==col){
		A_d[row * Awidth + col] += NoiseVec[row];
	}

}

__global__ void MatMulKernel(int Arow,int Acol,int Brow, int Bcol,double *A,double *B,double *C)
{

	int Crow=Arow;
	int Ccol=Bcol;
    double Ctemp = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    __syncthreads();

	if(row < Arow && col < Bcol) {

   		Ctemp = A[row] * B[col * Brow + row];
						  //GGTest[col*N + row]


	   C[col*Crow+row] = Ctemp;
	}
	  // 
}



extern "C" void WhiteMarginGPUWrapper_(double *Noise, double *Res, double *likeInfo, int N, int G)
{

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	
	double *GRes;
	GRes = (double*)malloc(sizeof(double)*G);
	double *WorkingGRes;
	WorkingGRes = (double*)malloc(sizeof(double)*G);

	// declare GPU copies
	double *Res_d;
	double *Noise_d;
	double *dettemp_d;
	
	double *NG_d;
	double *GG_d;
	double *GRes_d;



    hipError_t err;
    culaStatus status;

   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
 	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
	 
 	 err = hipMalloc( (void **)&NG_d, sizeof(double)*N*G);
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&GG_d, sizeof(double)*G*G);
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&GRes_d, sizeof(double)*G*G);
	 checkCudaError(err);


   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	 err = hipMemcpy(Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;

	 MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, G,Noise_d,GlobalGmat_d,NG_d);
 	
 	double alpha=1.0;
 	double beta=0.0; 
 	status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, NG_d, N, beta, GG_d, G);
	checkStatus(status);


 	 status = culaDeviceDpotrf('L', G, GG_d, G);
     checkStatus(status);

     calc_det<<< 1, 1 >>>( GG_d, dettemp_d, G);
     err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     likeInfo[0]=dettemp[0];
	
	 status = culaDeviceDgemv('T', N, G, alpha, GlobalGmat_d, N, Res_d, 1, beta, GRes_d, 1);
     checkStatus(status);
 	 err = hipMemcpy(GRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

 	 status=culaDeviceDpotrs('L', G, 1, GG_d, G, GRes_d, G);
	 checkStatus(status);
	 err = hipMemcpy(WorkingGRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
	 
	 double sum=0;
	 for(int i=0; i<G;i++){sum=sum+GRes[i]*WorkingGRes[i];}
	 likeInfo[1]=sum;
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
	 hipFree(Noise_d);
	 hipFree(NG_d);
	 hipFree(GG_d);
	 hipFree(GRes_d);

	 
  	 free(GRes); 
  	 free(dettemp);
  	 free(WorkingGRes);


   return;
}


extern "C" void vHRedMarginGPUWrapper_(double *Res, double *BatVec, double *NoiseVec, double *SpecInfo, double *likeInfo, double *FactorialList, int N, int G)
{


	hipError_t err;
	culaStatus status;
	
	double secday=24*60*60;
	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecInfo[1];
	double covconst=gsl_sf_gamma(1-modelalpha)*sin(0.5*M_PI*modelalpha);
	
	double gwamp=pow(10.0,SpecInfo[0]);
	double gwampsquared=gwamp*gwamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-modelalpha)))/(pow(flo,(modelalpha-1)));
	
	SpecInfo[0]=gwampsquared;
	SpecInfo[2]=covconst;
	
	double *Res_d;
	double *BatVec_d;
	double *NoiseVec_d;
	double *SpecInfo_d;
	double *CovMatrix_d;

  
  	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&BatVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&NoiseVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&SpecInfo_d, sizeof(double)*3 );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 

     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( BatVec_d, BatVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( NoiseVec_d, NoiseVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy(SpecInfo_d, SpecInfo, sizeof(double)*3, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;//((G + dimBlock.x - 1) / dimBlock.x,(N + dimBlock.y - 1) / dimBlock.y);
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
	 Makecov<<<dimGrid, dimBlock>>>(CovMatrix_d, BatVec_d, NoiseVec_d, SpecInfo_d, N,N);
	 

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	
	double *GRes;
	GRes =(double*)malloc(sizeof(double)*G);
	double *WorkingGRes;
	WorkingGRes =(double*)malloc(sizeof(double)*G);
	

	// declare GPU copies

	double *CG_d;
	double *GCG_d;
	double *GRes_d;
	double *dettemp_d;


   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);

	 
	  	 err = hipMalloc( (void **)&CG_d, sizeof(double)*N*G );
	 checkCudaError(err);
	  	 err = hipMalloc( (void **)&GCG_d, sizeof(double)*G*G );
	 checkCudaError(err);
	  	 err = hipMalloc( (void **)&GRes_d, sizeof(double)*G );
	 checkCudaError(err);

   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	// err = hipMemcpy(CovMatrix_d, CovMatrix, sizeof(double)*N*N, hipMemcpyHostToDevice );
 	// checkCudaError(err);

 	 
	int carryOn=0;
  	double alpha=1.0;
 	double beta=0.0; 

 	status =  culaDeviceDsymm('L', 'U', N, G, alpha, CovMatrix_d, N, GlobalGmat_d, N, beta, CG_d, N);
	checkStatus(status);
	//printf("done first linalg\n");
  	status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, CG_d, N, beta, GCG_d, G);
	checkStatus(status);

	 status = culaDeviceDgemv('T', N, G, alpha, GlobalGmat_d, N, Res_d, 1, beta, GRes_d, 1);
     	checkStatus(status);
 	 err = hipMemcpy(GRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);	 
 	 //	printf("do chol\n");
		
 	 status = culaDeviceDpotrf('L', G, GCG_d, G);
     	checkStatusCarryOn(status,carryOn);

	calc_det<<< 1, 1 >>>( GCG_d, dettemp_d, G);
	err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
	checkCudaError(err);
     	likeInfo[0]=dettemp[0];
    // printf("det: %g \n",likeInfo[0]);
    
	if(carryOn == 1){
     	// printf("Bad chol\n");
     	 
		hipFree(dettemp_d);
		hipFree(Res_d);
		hipFree(BatVec_d);
		hipFree(NoiseVec_d);
		hipFree(SpecInfo_d);
		hipFree(CovMatrix_d);
		hipFree(CG_d);
		hipFree(GCG_d);
		hipFree(GRes_d);
	
		free(dettemp);
		free(GRes);
		free(WorkingGRes);
		
		
		return;

  	 }

 	 status=culaDeviceDpotrs('L', G, 1, GCG_d, G, GRes_d, G);
	 checkStatus(status);
	 err = hipMemcpy(WorkingGRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

	 double sum=0;
	 for(int i=0; i<G;i++){sum=sum+GRes[i]*WorkingGRes[i];}
	 likeInfo[1]=sum;
	 //printf("like: %g \n",sum);
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
  	 hipFree(BatVec_d);
 	 hipFree(NoiseVec_d);
 	 hipFree(SpecInfo_d);
	 hipFree(CovMatrix_d);
 	 hipFree(CG_d);
	 hipFree(GCG_d);
	 hipFree(GRes_d);

  	 free(dettemp);
   	 free(GRes);
  	 free(WorkingGRes);
  


   return;
}

extern "C" void vHRedGPUWrapper_(double *SpecInfo, double *BatVec, double *Res, double *NoiseVec, double *likeInfo, int N)
{

	hipError_t err;
	culaStatus status;

	double secday=24*60*60;
	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecInfo[1];
	double covconst=gsl_sf_gamma(1-modelalpha)*sin(0.5*M_PI*modelalpha);
	
	double gwamp=pow(10.0,SpecInfo[0]);
	double gwampsquared=gwamp*gwamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-modelalpha)))/(pow(flo,(modelalpha-1)));
	
	SpecInfo[0]=gwampsquared;
	SpecInfo[2]=covconst;
	
	double *Res_d;
	double *BatVec_d;
	double *NoiseVec_d;
	double *SpecInfo_d;
	double *CovMatrix_d;

  
  	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&BatVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&NoiseVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&SpecInfo_d, sizeof(double)*3 );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 

     	 err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( BatVec_d, BatVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( NoiseVec_d, NoiseVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy(SpecInfo_d, SpecInfo, sizeof(double)*3, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;//((G + dimBlock.x - 1) / dimBlock.x,(N + dimBlock.y - 1) / dimBlock.y);
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
	 Makecov<<<dimGrid, dimBlock>>>(CovMatrix_d, BatVec_d, NoiseVec_d, SpecInfo_d, N,N);
	 

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	

	double *WorkingRes;
	WorkingRes = (double*)malloc(sizeof(double)*N);

	// declare GPU copies
	double *dettemp_d;



   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);

   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);

	 int carryOn=0;
 	 status = culaDeviceDpotrf('L', N, CovMatrix_d, N);
     	 checkStatusCarryOn(status,carryOn);
     


	 calc_det<<< 1, 1 >>>( CovMatrix_d, dettemp_d, N);
	 err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     	 likeInfo[0]=dettemp[0];
     
          if(carryOn == 1){

     	 	 hipFree(dettemp_d);
	 	 hipFree(Res_d);
		 hipFree(BatVec_d);
		 hipFree(NoiseVec_d);
		 hipFree(SpecInfo_d);
		 hipFree(CovMatrix_d);
	  	 free(dettemp);
	  	 free(WorkingRes);
	  	 return;
  	 }
	

 	 status=culaDeviceDpotrs('L', N, 1, CovMatrix_d, N, Res_d, N);
	 checkStatus(status);
	 err = hipMemcpy(WorkingRes, Res_d, sizeof(double)*N, hipMemcpyDeviceToHost);

  	 checkCudaError(err);

	 double sum=0;
	 	
	for(int i=0; i<N;i++){
		sum=sum+Res[i]*WorkingRes[i];
	}

	 likeInfo[1]=sum;
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
  	 hipFree(BatVec_d);
 	 hipFree(NoiseVec_d);
 	 hipFree(SpecInfo_d);
	 hipFree(CovMatrix_d);

  	 free(dettemp);
  	 free(WorkingRes);
  	 
  


   return;
}


// simple kernel function that calculates the FMatrix
__global__ void make_fmatrix(double *FMatrix_d,double *Freqs_d, double *BATvec_d, int N,int F)
{

	int Bidx = blockIdx.x;
	
	for(int i=0;i<F/2;i++){
		
			FMatrix_d[i*N + Bidx]=cos(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
			FMatrix_d[(i+F/2)*N + Bidx]=sin(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
	}


}
__global__ void fastmake_fmatrix(double *FMatrix_d,double *Freqs_d, double *BATvec_d, int Aheight,int Awidth) {

	// Each thread computes one element of F
	// by accumulating results into Cvalue


	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;

	FMatrix_d[row * Awidth + col]=cos(2*M_PI*Freqs_d[col]*BATvec_d[row]);
	FMatrix_d[row*Awidth + col + Awidth/2]=sin(2*M_PI*Freqs_d[col]*BATvec_d[row]);

}



extern "C" void LRedGPUWrapper_(double *Freqs, double *resvec, double *BATvec, double *Noise, double **FNF, double *NFd, int N, int F){

	double *Freqs_d;
	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	
	double *FMatrix_d;
	double *NF_d;	
	double *FNF_d;
	double *NFd_d;
	
	double *FNFvec;
	FNFvec = (double*)malloc(sizeof(double)*F*F);

	hipError_t err;
	culaStatus status;
      
  	 err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
	 
   	 err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&NF_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&FNF_d, sizeof(double)*F*F );
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&NFd_d, sizeof(double)*F);
	 checkCudaError(err);	
	 
         // copy vectors from CPU to GPU
   	 err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 
// 	 make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,N,F);
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;

	 dimGrid.x=(F + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 fastmake_fmatrix<<<dimGrid, dimBlock>>>(FMatrix_d,Freqs_d,BATvec_d,N,F);

	 MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, F,Noise_d,FMatrix_d,NF_d);

 	 
	double alpha=1.0;
	double beta=0.0; 
	status =  culaDeviceDgemm('T', 'N', F, F, N, alpha, FMatrix_d, N, NF_d, N, beta, FNF_d, F);
	checkStatus(status);
	
 	 status = culaDeviceDgemv('T', N, F, alpha, NF_d, N, resvec_d, 1, beta, NFd_d, 1);
	 checkStatus(status);
	
	 err = hipMemcpy(FNFvec, FNF_d, sizeof(double)*F*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 	
	 err = hipMemcpy(NFd, NFd_d, sizeof(double)*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

	for(int f1=0;f1<F; f1++){
		for(int f2=0;f2<F; f2++){

			FNF[f2][f1]=FNFvec[f1*F + f2];
		}
	}

	hipFree(Freqs_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
    	hipFree(FMatrix_d);
    	hipFree(NF_d);
	hipFree(FNF_d);
	hipFree(resvec_d);
	hipFree(NFd_d);
	
	free(FNFvec);
		
}
 	 


extern "C" void LRedMarginGPUWrapper_(double *Freqs, double *resvec, double *BATvec, double *Noise, double **FNF, double *NFd, double *likeVals, int N, int F, int G){

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	double *dettemp_d;

	double *Freqs_d;
	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	
		
	double *NG_d;
	double *GG_d;
	double *GNG_d;
	double *GNGd_d;
	
	double *GNGd;
	GNGd = (double*)malloc(sizeof(double)*N);

	
	double *FMatrix_d;
	double *NF_d;	
	double *FNF_d;
	double *NFd_d;
	
	double *FNFvec;
	FNFvec = (double*)malloc(sizeof(double)*F*F);

	hipError_t err;
	culaStatus status;
    
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
      
  	 err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GNGd_d, sizeof(double)*N );
	 checkCudaError(err);

	 
   	 err = hipMalloc( (void **)&NG_d, sizeof(double)*N*G);
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GG_d, sizeof(double)*G*G );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&GNG_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 
   	 err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&NF_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&FNF_d, sizeof(double)*F*F );
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&NFd_d, sizeof(double)*F);
	 checkCudaError(err);	
	 
   // copy vectors from CPU to GPU
   	 err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;

	 MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, G,Noise_d,GlobalGmat_d,NG_d);
	 

 	 double alpha=1.0;
 	 double beta=0.0; 
 	 status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, NG_d, N, beta, GG_d, G);
	 checkStatus(status);


 	 status = culaDeviceDpotrf('L', G, GG_d, G);
     	checkStatus(status);

	calc_det<<< 1, 1 >>>( GG_d, dettemp_d, G);
	err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     	likeVals[0]=dettemp[0];
     
  	 status = culaDeviceDpotri('L', G, GG_d, G);
     	checkStatus(status);
     
  	 status =  culaDeviceDsymm('R', 'L', N, G, alpha, GG_d, G, GlobalGmat_d, N, beta, NG_d, N);
	 checkStatus(status);
	 
  	 status =  culaDeviceDgemm('N', 'T', N, N, G, alpha, NG_d, N, GlobalGmat_d, N, beta, GNG_d, N);
	 checkStatus(status);
	 
  	 status = culaDeviceDgemv('N', N, N, alpha, GNG_d, N, resvec_d, 1, beta, GNGd_d, 1);
     	checkStatus(status);
     

	 err = hipMemcpy(GNGd, GNGd_d, sizeof(double)*N, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 likeVals[1]=0;
  	 for(int i =0;i < N; i++){likeVals[1] += resvec[i]*GNGd[i];}

 	 make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,N,F);
 	 
   	 status =  culaDeviceDgemm('N', 'N', N, F, N, alpha, GNG_d, N, FMatrix_d, N, beta, NF_d, N);
	 checkStatus(status);
	 
  	 status =  culaDeviceDgemm('T', 'N', F, F, N, alpha, FMatrix_d, N, NF_d, N, beta, FNF_d, F);
	 checkStatus(status);
	
 	 status = culaDeviceDgemv('T', N, F, alpha, NF_d, N, resvec_d, 1, beta, NFd_d, 1);
     checkStatus(status);
	
	 err = hipMemcpy(FNFvec, FNF_d, sizeof(double)*F*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 	
	 err = hipMemcpy(NFd, NFd_d, sizeof(double)*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

		for(int f1=0;f1<F; f1++){
			for(int f2=0;f2<F; f2++){
 
				FNF[f2][f1]=FNFvec[f1*F + f2];
				//printf("GPUFNF: %i %i %g \n",f1,f2);
			}
		}

	hipFree(dettemp_d);
	hipFree(Freqs_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
    hipFree(FMatrix_d);
    hipFree(NF_d);
	hipFree(FNF_d);
	hipFree(resvec_d);
	hipFree(NFd_d);
	
    hipFree(NG_d);
    hipFree(GG_d);
	hipFree(GNG_d);
	hipFree(GNGd_d);

	
	free(FNFvec);
	free(dettemp);
	free(GNGd);
	
}
 	 

 extern "C" void copy_floatgmat_(float *G, int N){

    hipError_t err;

   // Allocate memory on GPU
	printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalGmatFloat_d, sizeof(float)*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalGmatFloat_d, G, sizeof(float)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}


extern "C" void copy_gmat_(double *G, int N){

    hipError_t err;

   // Allocate memory on GPU
	printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalGmat_d, sizeof(double)*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalGmat_d, G, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}

