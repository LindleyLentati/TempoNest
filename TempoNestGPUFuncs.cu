#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <gsl/gsl_sf_gamma.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cula_lapack_device.h>
#include <cula_blas_device.h>


#define BLOCK_SIZE 16

double *GlobalGmat_d;
double *GlobalStaticGmat_d;
double *GlobalStaticUGmat_d;
float *GlobalGmatFloat_d;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)

double iter_factorialGPU(unsigned int n)
{
    double ret = 1;
    for(unsigned int i = 1; i <= n; ++i)
        ret *= i;
    return ret;
}


void checkStatus(culaStatus status)
{
    char buf[256];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    culaShutdown();
    exit(EXIT_FAILURE);
}

void checkStatusCarryOn(culaStatus status, int &CarryOn)
{
    char buf[256];

    if(!status){
    	CarryOn=0;
        return;
        }

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    //printf("%s\n", buf);
    CarryOn=1;
    
    culaShutdown();
    culaStatus status2;
	status2 = culaInitialize();
	
}


void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    culaShutdown();
    exit(EXIT_FAILURE);
}


// simple kernel function that adds two vectors
__global__ void vect_add(double *a, double *b, int N)
{
   int Bidx = blockIdx.x;
   //a[Bidx*N+Tidx] = a[Bidx*N+Tidx] + b[Bidx*N+Tidx]; 
   for(int i =0; i < N; i++){
   		a[Bidx*N+i] = a[Bidx*N+i] + b[Bidx*N+i]; 
   		}
}


// simple kernel function that calcs det of a matrix
__global__ void calc_det(double *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void Floatcalc_det(float *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void calc_detFloat(float *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void calc_DiagLike(double *Vec, double *Noise, int N, double *val)
{
	
	val[0]=0;
	for(int i =0; i < N; i++){
   		val[0]+=Vec[i]*Vec[i]*Noise[i];
	}

   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void calc_DotLike(double *Vec1, double *Vec2, int N, double *val)
{
	
	val[0]=0;
	for(int i =0; i < N; i++){
   		val[0]+=Vec1[i]*Vec2[i];
	}

   		
   		
}


__global__ void Makecov(double *A_d, double *BATvec, double *NoiseVec, double *SpecParm, int Aheight, int Awidth) {

	// Each thread computes one element of C
	// by accumulating results into Cvalue
	

	double LongestPeriod=1.0/pow(10.0,-5); //
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecParm[1];
	double gwampsquared=SpecParm[0];
	double covconst=SpecParm[2];
	

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;
	double timdiff= BATvec[row] - BATvec[col];	
	double tau=2.0*M_PI*fabs(timdiff);
	double covsum=0;

	for(int k=0; k <=10; k++){
	
		    double ret = 1;
			for(unsigned int i = 1; i <= 2*k; ++i){
				ret *= (double)i;
			//	printf("Ret: %i %g \n",i,ret);
			}
    
			covsum=covsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(ret*(2*k+1-modelalpha));
			//printf("covsum: %i %i %i %g \n",row,col,k,covsum);

	}

	A_d[row * Awidth + col]=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);

	if(row==col){
		A_d[row * Awidth + col] += NoiseVec[row];
	}

}


__global__ void MakeDMcov(double *A_d, double *BATvec, double *NoiseVec, double *DMVec, double *SpecParm, int Aheight, int Awidth) {

	// Each thread computes one element of C
	// by accumulating results into Cvalue
	

	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double gwampsquared=SpecParm[0];
	double modelalpha=SpecParm[1];
	double covconst=SpecParm[2];
	
	double dmampsquared=SpecParm[3];
	double dmmodelalpha=SpecParm[4];
	double dmcovconst=SpecParm[5];
	

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;
	double timdiff= BATvec[row] - BATvec[col];	
	double tau=2.0*M_PI*fabs(timdiff);
	double covsum=0;
	double dmcovsum=0;

	for(int k=0; k <=5; k++){
	
		    double ret = 1;
			for(unsigned int i = 1; i <= 2*k; ++i){
				ret *= (double)i;
			}
    
			covsum=covsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(ret*(2*k+1-modelalpha));
			
			dmcovsum=dmcovsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(ret*(2*k+1-dmmodelalpha));

	}
	
	double gwpart=0;
	if(SpecParm[0] !=0 )gwpart=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);
	
	double dmpart=0;
	if(SpecParm[3] !=0 )dmpart=dmampsquared*(dmcovconst*pow((flo*tau),(dmmodelalpha-1)) - dmcovsum)*DMVec[row]*DMVec[col];

	A_d[row * Awidth + col]= gwpart+dmpart;

	if(row==col){
		A_d[row * Awidth + col] += NoiseVec[row];
	}

	//printf("%i %i %g\n",row,col,A_d[row * Awidth + col]);

}


__global__ void MatMulKernel(int Arow,int Acol,int Brow, int Bcol,double *A,double *B,double *C)
{

	int Crow=Arow;
	int Ccol=Bcol;
    double Ctemp = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    __syncthreads();

	if(row < Arow && col < Bcol) {
		//if(row==0)printf("NG: %i %i %g %g \n", row, col, B[col * Brow + row], A[row] );
   		Ctemp = A[row] * B[col * Brow + row];
						  //GGTest[col*N + row]


	   C[col*Crow+row] = Ctemp;
	}
	  // 
}



extern "C" void WhiteMarginGPUWrapper_(double *Noise, double *Res, double *likeInfo, int N, int G, int incEFAC, int incEQUAD)
{

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	
	double *GRes;
	GRes = (double*)malloc(sizeof(double)*G);
	double *WorkingGRes;
	WorkingGRes = (double*)malloc(sizeof(double)*G);

	// declare GPU copies
	double *Res_d;
	double *Noise_d;
	double *dettemp_d;
	
	double *NG_d;
	double *GG_d;
	double *GRes_d;



    hipError_t err;
    culaStatus status;

   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
 	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
	 
 	 err = hipMalloc( (void **)&NG_d, sizeof(double)*N*G);
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&GG_d, sizeof(double)*G*G);
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&GRes_d, sizeof(double)*G*G);
	 checkCudaError(err);


   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
         err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 
 	 

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid;
	dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
 	double alpha=1.0;
 	double beta=0.0; 
	 
	 if(incEFAC >1 || incEQUAD >1){

		 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
		 checkCudaError(err);
		 err = hipMemcpy(Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
		 checkCudaError(err);


		 MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, G,Noise_d,GlobalGmat_d,NG_d);
	 	

	 	status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, NG_d, N, beta, GG_d, G);
		checkStatus(status);


	 	 status = culaDeviceDpotrf('L', G, GG_d, G);
		 checkStatus(status);

		 calc_det<<< 1, 1 >>>( GG_d, dettemp_d, G);
		 err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
	  	 checkCudaError(err);
		 likeInfo[0]=dettemp[0];
	
		 status = culaDeviceDgemv('T', N, G, alpha, GlobalGmat_d, N, Res_d, 1, beta, GRes_d, 1);
		 checkStatus(status);
	 	 err = hipMemcpy(GRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
	  	 checkCudaError(err);

	 	 status=culaDeviceDpotrs('L', G, 1, GG_d, G, GRes_d, G);
		 checkStatus(status);
		 err = hipMemcpy(WorkingGRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
	  	 checkCudaError(err);
		 
		 double sum=0;
		 for(int i=0; i<G;i++){sum=sum+GRes[i]*WorkingGRes[i];}
		 likeInfo[1]=sum;

		hipFree(Noise_d);
	}
	else if(incEFAC == 1 || incEQUAD==1 &&incEFAC <2 && incEQUAD<2){

                 err = hipMalloc( (void **)&Noise_d, sizeof(double)*G );
                 checkCudaError(err);
                 err = hipMemcpy(Noise_d, Noise, sizeof(double)*G, hipMemcpyHostToDevice );
                 checkCudaError(err);
	
		status = culaDeviceDgemv('N', G, N, alpha, GlobalStaticUGmat_d, G, Res_d, 1, beta, GRes_d, 1);
		checkStatus(status);
		
		calc_DiagLike<<< 1, 1 >>>(GRes_d, Noise_d, G, dettemp_d);
		err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
		checkCudaError(err);
		likeInfo[1]=dettemp[0];

		hipFree(Noise_d);
	}
	else if(incEFAC == 0 && incEQUAD == 0){
	
		double *NRes_d;
		err = hipMalloc( (void **)&NRes_d, sizeof(double)*N );
		checkCudaError(err);
		status = culaDeviceDgemv('T', N, N, alpha, GlobalStaticGmat_d, N, Res_d, 1, beta, NRes_d, 1);
     	checkStatus(status);

		calc_DotLike<<< 1, 1 >>>(NRes_d, Res_d, N, dettemp_d);
		
     	err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
		checkCudaError(err);
		likeInfo[1]=dettemp[0];
				
		hipFree(NRes_d);
	}
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
	 hipFree(NG_d);
	 hipFree(GG_d);
	 hipFree(GRes_d);

	 
  	 free(GRes); 
  	 free(dettemp);
  	 free(WorkingGRes);


   return;
}


extern "C" void vHRedMarginGPUWrapper_(double *Res, double *BatVec, double *DMVec, double *NoiseVec, double *SpecInfo, double *likeInfo, double *FactorialList, int N, int G)
{


	hipError_t err;
	culaStatus status;
	
	double secday=24*60*60;
	double LongestPeriod= 1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double redalpha=0;
	double redcovconst=0;
	double redamp=0;
	double redampsquared=0;
	double DMalpha=0;
	double DMcovconst=0;
	double DMamp=0;
	double DMampsquared=0;	
	
	
	
	if(SpecInfo[0] != 0){

		redalpha=SpecInfo[1];
		redamp=pow(10.0,SpecInfo[0]);
		redampsquared=redamp*redamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-redalpha)))/(pow(flo,(redalpha-1)));
		//redampsquared=redamp*redamp*(pow(365.25,(1-redalpha)))/(pow(flo,(redalpha-1)));
		redcovconst=gsl_sf_gamma(1-redalpha)*sin(0.5*M_PI*redalpha);
	
		SpecInfo[0]=redampsquared;
		SpecInfo[2]=redcovconst;

	}
	
	 double *DMVec_d;
	 err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
	 checkCudaError(err);
	
	if(SpecInfo[3] != 0){
	

		DMalpha=SpecInfo[4];
		DMamp=pow(10.0,SpecInfo[3]);
		DMampsquared=DMamp*DMamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-DMalpha)))/(pow(flo,(DMalpha-1)));
		//DMampsquared=DMamp*DMamp*(pow(365.25,(1-DMalpha)))/(pow(flo,(DMalpha-1)));
		DMcovconst=gsl_sf_gamma(1-DMalpha)*sin(0.5*M_PI*DMalpha);
	
		SpecInfo[3]=DMampsquared;
		SpecInfo[5]=DMcovconst;	
		
        err = hipMemcpy( DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	    checkCudaError(err);
	}
	

	
	double *Res_d;
	double *BatVec_d;
	double *NoiseVec_d;
	double *SpecInfo_d;
	double *CovMatrix_d;

  
  	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&BatVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&NoiseVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&SpecInfo_d, sizeof(double)*6 );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 

     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( BatVec_d, BatVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( NoiseVec_d, NoiseVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy(SpecInfo_d, SpecInfo, sizeof(double)*6, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;//((G + dimBlock.x - 1) / dimBlock.x,(N + dimBlock.y - 1) / dimBlock.y);
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
	 MakeDMcov<<<dimGrid, dimBlock>>>(CovMatrix_d, BatVec_d, NoiseVec_d,  DMVec_d, SpecInfo_d, N,N);


	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	
	double *GRes;
	GRes =(double*)malloc(sizeof(double)*G);
	double *WorkingGRes;
	WorkingGRes =(double*)malloc(sizeof(double)*G);
	

	// declare GPU copies

	double *CG_d;
	double *GCG_d;
	double *GRes_d;
	double *dettemp_d;


   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);

	 
	  	 err = hipMalloc( (void **)&CG_d, sizeof(double)*N*G );
	 checkCudaError(err);
	  	 err = hipMalloc( (void **)&GCG_d, sizeof(double)*G*G );
	 checkCudaError(err);
	  	 err = hipMalloc( (void **)&GRes_d, sizeof(double)*G );
	 checkCudaError(err);

   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	// err = hipMemcpy(CovMatrix_d, CovMatrix, sizeof(double)*N*N, hipMemcpyHostToDevice );
 	// checkCudaError(err);

 	 
	int carryOn=0;
  	double alpha=1.0;
 	double beta=0.0; 

 	status =  culaDeviceDsymm('L', 'U', N, G, alpha, CovMatrix_d, N, GlobalGmat_d, N, beta, CG_d, N);
	checkStatus(status);
	//printf("done first linalg\n");
  	status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, CG_d, N, beta, GCG_d, G);
	checkStatus(status);

	 status = culaDeviceDgemv('T', N, G, alpha, GlobalGmat_d, N, Res_d, 1, beta, GRes_d, 1);
     	checkStatus(status);
 	 err = hipMemcpy(GRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);	 
 	 //	printf("do chol\n");
		
 	 status = culaDeviceDpotrf('L', G, GCG_d, G);
     	checkStatusCarryOn(status,carryOn);

	calc_det<<< 1, 1 >>>( GCG_d, dettemp_d, G);
	err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
	checkCudaError(err);
     	likeInfo[0]=dettemp[0];
    // printf("det: %g \n",likeInfo[0]);
    
	if(carryOn == 1){
     	// printf("Bad chol\n");
     	 
		hipFree(dettemp_d);
		hipFree(Res_d);
		hipFree(BatVec_d);
		hipFree(NoiseVec_d);
		hipFree(SpecInfo_d);
		hipFree(CovMatrix_d);
		hipFree(CG_d);
		hipFree(GCG_d);
		hipFree(GRes_d);
		hipFree(DMVec_d);
	
		free(dettemp);
		free(GRes);
		free(WorkingGRes);
		
		
		return;

  	 }

 	 status=culaDeviceDpotrs('L', G, 1, GCG_d, G, GRes_d, G);
	 checkStatus(status);
	 err = hipMemcpy(WorkingGRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

	 double sum=0;
	 for(int i=0; i<G;i++){sum=sum+GRes[i]*WorkingGRes[i];}
	 likeInfo[1]=sum;
	 //printf("like: %g \n",sum);
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
  	 hipFree(BatVec_d);
 	 hipFree(NoiseVec_d);
 	 hipFree(SpecInfo_d);
	 hipFree(CovMatrix_d);
 	 hipFree(CG_d);
	 hipFree(GCG_d);
	 hipFree(GRes_d);
	 hipFree(DMVec_d);

  	 free(dettemp);
   	 free(GRes);
  	 free(WorkingGRes);
  


   return;
}

extern "C" void vHRedGPUWrapper_(double *SpecInfo, double *BatVec,  double *DMVec, double *Res, double *NoiseVec, double *likeInfo, int N)
{

	hipError_t err;
	culaStatus status;
	
	
	double secday=24*60*60;
	double LongestPeriod= 1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double redalpha=0;
	double redcovconst=0;
	double redamp=0;
	double redampsquared=0;
	double DMalpha=0;
	double DMcovconst=0;
	double DMamp=0;
	double DMampsquared=0;	
	

	if(SpecInfo[0] != 0){
	
		redalpha=SpecInfo[1];
		redamp=pow(10.0,SpecInfo[0]);
		redampsquared=redamp*redamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-redalpha)))/(pow(flo,(redalpha-1)));
		redcovconst=gsl_sf_gamma(1-redalpha)*sin(0.5*M_PI*redalpha);
	
		SpecInfo[0]=redampsquared;
		SpecInfo[2]=redcovconst;

	}
	
	 double *DMVec_d;
	 err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
	 checkCudaError(err);
	if(SpecInfo[3] != 0){
	
		DMalpha=SpecInfo[4];
		DMamp=pow(10.0,SpecInfo[3]);
		DMampsquared=DMamp*DMamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-DMalpha)))/(pow(flo,(DMalpha-1)));
		DMcovconst=gsl_sf_gamma(1-DMalpha)*sin(0.5*M_PI*DMalpha);
	
		SpecInfo[3]=DMampsquared;
		SpecInfo[5]=DMcovconst;	
		
        err = hipMemcpy( DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	    checkCudaError(err);
	}
	
	
	double *Res_d;
	double *BatVec_d;
	double *NoiseVec_d;
	double *SpecInfo_d;
	double *CovMatrix_d;

  
  	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&BatVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&NoiseVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&SpecInfo_d, sizeof(double)*6 );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 

     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( BatVec_d, BatVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( NoiseVec_d, NoiseVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy(SpecInfo_d, SpecInfo, sizeof(double)*6, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;//((G + dimBlock.x - 1) / dimBlock.x,(N + dimBlock.y - 1) / dimBlock.y);
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
	 MakeDMcov<<<dimGrid, dimBlock>>>(CovMatrix_d, BatVec_d, NoiseVec_d,  DMVec_d, SpecInfo_d, N,N);
	 

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	

	double *WorkingRes;
	WorkingRes = (double*)malloc(sizeof(double)*N);

	// declare GPU copies
	double *dettemp_d;



   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);

   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);

	 int carryOn=0;
 	 status = culaDeviceDpotrf('L', N, CovMatrix_d, N);
     	 checkStatusCarryOn(status,carryOn);
     


	 calc_det<<< 1, 1 >>>( CovMatrix_d, dettemp_d, N);
	 err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     	 likeInfo[0]=dettemp[0];
     
          if(carryOn == 1){

     	 	 hipFree(dettemp_d);
	 	 hipFree(Res_d);
		 hipFree(BatVec_d);
		 hipFree(NoiseVec_d);
		 hipFree(SpecInfo_d);
		 hipFree(CovMatrix_d);
	  	 free(dettemp);
	  	 free(WorkingRes);
	  	 hipFree(DMVec_d);
	  	 return;
  	 }
	

 	 status=culaDeviceDpotrs('L', N, 1, CovMatrix_d, N, Res_d, N);
	 checkStatus(status);
	 err = hipMemcpy(WorkingRes, Res_d, sizeof(double)*N, hipMemcpyDeviceToHost);

  	 checkCudaError(err);

	 double sum=0;
	 	
	for(int i=0; i<N;i++){
		sum=sum+Res[i]*WorkingRes[i];

	}

	 likeInfo[1]=sum;
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
  	 hipFree(BatVec_d);
 	 hipFree(NoiseVec_d);
 	 hipFree(SpecInfo_d);
	 hipFree(CovMatrix_d);
	 hipFree(DMVec_d);

  	 free(dettemp);
  	 free(WorkingRes);
  	 
  


   return;
}


// simple kernel function that calculates the FMatrix
__global__ void make_fmatrix(double *FMatrix_d,double *Freqs_d, double *BATvec_d, double *DMVec_d, int N,int RF, int DMF, int incRED, int incDM)
{

	int Bidx = blockIdx.x;
	int startpos=0;
	if(incRED !=0){
		for(int i=0;i<RF/2;i++){
		//	if(Bidx==0)printf("FM: %i %g %g \n",i,Freqs_d[i], BATvec_d[Bidx]);	
			FMatrix_d[i*N + Bidx]=cos(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
			FMatrix_d[(i+RF/2)*N + Bidx]=sin(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
		}
		startpos=RF;
	}

	
      if(incDM !=0){
                for(int i=0;i<DMF/2;i++){
			//if(Bidx==0)printf("D: %i %i %g %g \n", Bidx,i,1.0/Freqs_d[i], DMVec_d[Bidx]);
                        FMatrix_d[(startpos+i)*N + Bidx]=cos(2*M_PI*Freqs_d[startpos+i]*BATvec_d[Bidx])*DMVec_d[Bidx];
                        FMatrix_d[(startpos+i+DMF/2)*N + Bidx]=sin(2*M_PI*Freqs_d[startpos+i]*BATvec_d[Bidx])*DMVec_d[Bidx];
                }
       }

}


// simple kernel function that calculates the FMatrix
__global__ void make_DMfmatrix(double *FMatrix_d,double *Freqs_d, double *BATvec_d, double *DMVec_d, int N,int F)
{

	int Bidx = blockIdx.x;
	
	for(int i=0;i<F/4;i++){
		
			FMatrix_d[i*N + Bidx]=cos(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
			FMatrix_d[(i+F/4)*N  + Bidx]=cos(2*M_PI*Freqs_d[i]*BATvec_d[Bidx])*DMVec_d[Bidx];
			FMatrix_d[(i+F/2)*N + Bidx]=sin(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
			FMatrix_d[(i+3*F/4)*N + Bidx]=sin(2*M_PI*Freqs_d[i]*BATvec_d[Bidx])*DMVec_d[Bidx];
	}


}


__global__ void fastmake_fmatrix(double *FMatrix_d,double *Freqs_d, double *BATvec_d, int Aheight,int Awidth) {

	// Each thread computes one element of F
	// by accumulating results into Cvalue


	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;

	FMatrix_d[row * Awidth + col]=cos(2*M_PI*Freqs_d[col]*BATvec_d[row]);
	FMatrix_d[row*Awidth + col + Awidth/2]=sin(2*M_PI*Freqs_d[col]*BATvec_d[row]);

}




extern "C" void LRedGPUWrapper_(double *Freqs, double *resvec, double *BATvec, double *DMVec, double *Noise, double **FNF, double *NFd, int N, int RF,int DMF, int F, int incRED, int incDM){

	double *Freqs_d;
	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	double *DMVec_d;
	
	double *FMatrix_d;
	double *NF_d;	
	double *FNF_d;
	double *NFd_d;
	
	double *FNFvec;
	FNFvec = (double*)malloc(sizeof(double)*F*F);

	hipError_t err;
	culaStatus status;
      
  	 err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
         err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
         checkCudaError(err);

	 
   	 err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&NF_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&FNF_d, sizeof(double)*F*F );
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&NFd_d, sizeof(double)*F);
	 checkCudaError(err);	
	 
         // copy vectors from CPU to GPU
   	 err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
     err = hipMemcpy( DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
     checkCudaError(err);

 	 
// 	 make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,N,F);
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;

	 dimGrid.x=(F + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 //fastmake_fmatrix<<<dimGrid, dimBlock>>>(FMatrix_d,Freqs_d,BATvec_d,N,F);
 	 make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,DMVec_d,N,RF,DMF, incRED, incDM);

	 MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, F,Noise_d,FMatrix_d,NF_d);

 	 
	double alpha=1.0;
	double beta=0.0; 
	status =  culaDeviceDgemm('T', 'N', F, F, N, alpha, FMatrix_d, N, NF_d, N, beta, FNF_d, F);
	checkStatus(status);
	
 	 status = culaDeviceDgemv('T', N, F, alpha, NF_d, N, resvec_d, 1, beta, NFd_d, 1);
	 checkStatus(status);
	
	 err = hipMemcpy(FNFvec, FNF_d, sizeof(double)*F*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 	
	 err = hipMemcpy(NFd, NFd_d, sizeof(double)*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

	for(int f1=0;f1<F; f1++){
		for(int f2=0;f2<F; f2++){

			FNF[f2][f1]=FNFvec[f1*F + f2];
		}
	}

	hipFree(Freqs_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
    	hipFree(FMatrix_d);
    	hipFree(NF_d);
	hipFree(FNF_d);
	hipFree(resvec_d);
	hipFree(NFd_d);
	hipFree(DMVec_d);
	free(FNFvec);
		
}
 	 
 	 



extern "C" void LRedMarginGPUWrapper_(double *Freqs, double *resvec, double *BATvec, double *DMVec, double *Noise, double **FNF, double *NFd, double *likeVals, int N, int RF,int DMF, int F, int G, int incRED, int incDM, int incEFAC, int incEQUAD){

	//printf("%i %i \n", SF,F);
	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	double *dettemp_d;

	double *Freqs_d;
	double *resvec_d;
	double *BATvec_d;
	double *DMVec_d;
	double *Noise_d;
	
		
	double *NG_d;
	double *GG_d;
	double *GNG_d;
	double *GNGd_d;
	
	double *GNGd;
	GNGd = (double*)malloc(sizeof(double)*N);

	
	double *FMatrix_d;
	double *NF_d;	
	double *FNF_d;
	double *NFd_d;
	
	double *FNFvec;
	FNFvec = (double*)malloc(sizeof(double)*F*F);

	hipError_t err;
	culaStatus status;
    
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
      
  	 err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);
 	 err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
         checkCudaError(err);
	 err = hipMalloc( (void **)&GNGd_d, sizeof(double)*N );
	 checkCudaError(err);

	 
   	 err = hipMalloc( (void **)&NG_d, sizeof(double)*N*G);
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GG_d, sizeof(double)*G*G );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&GNG_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 
   	 err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&NF_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&FNF_d, sizeof(double)*F*F );
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&NFd_d, sizeof(double)*F);
	 checkCudaError(err);	
	 
   // copy vectors from CPU to GPU
   	 err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
         checkCudaError(err);

 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
  	 double alpha=1.0;
 	 double beta=0.0; 
	 
	 if(incEFAC == 0 && incEQUAD == 0){
	 
		status = culaDeviceDgemv('N', N, N, alpha, GlobalStaticGmat_d, N, resvec_d, 1, beta, GNGd_d, 1);
		checkStatus(status);
		err = hipMemcpy(GNGd, GNGd_d, sizeof(double)*N, hipMemcpyDeviceToHost);
		checkCudaError(err);
		likeVals[1]=0;
		likeVals[0]=0;
		for(int i =0;i < N; i++){likeVals[1] += resvec[i]*GNGd[i]; }
		
		make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,DMVec_d,N,RF, DMF, incRED, incDM);

		status =  culaDeviceDgemm('N', 'N', N, F, N, alpha, GlobalStaticGmat_d, N, FMatrix_d, N, beta, NF_d, N);
		checkStatus(status);
		
	}
	else if(incEFAC == 1 || incEQUAD==1 &&incEFAC < 2 && incEQUAD < 2){

	         err = hipMalloc( (void **)&Noise_d, sizeof(double)*G );
	         checkCudaError(err);
	         err = hipMemcpy( Noise_d, Noise, sizeof(double)*G, hipMemcpyHostToDevice );
	         checkCudaError(err);

	
		//printf("1\n");
		MatMulKernel<<<dimGrid, dimBlock>>>(G,G,G, N,Noise_d,GlobalStaticUGmat_d,NG_d);
		//printf("2\n");
		status =  culaDeviceDgemm('T', 'N', N, N, G, alpha, GlobalStaticUGmat_d, G, NG_d, G, beta, GNG_d, N);
		checkStatus(status);
		//printf("3\n");
		status = culaDeviceDgemv('N', N, N, alpha, GNG_d, N, resvec_d, 1, beta, GNGd_d, 1);
		checkStatus(status);
     	//printf("4\n");

		err = hipMemcpy(GNGd, GNGd_d, sizeof(double)*N, hipMemcpyDeviceToHost);
		checkCudaError(err);
		likeVals[1]=0;
		for(int i =0;i < N; i++){likeVals[1] += resvec[i]*GNGd[i]; }
		
		make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,DMVec_d,N,RF,DMF, incRED, incDM);

		status =  culaDeviceDgemm('N', 'N', N, F, N, alpha, GNG_d, N, FMatrix_d, N, beta, NF_d, N);
		checkStatus(status);
		//printf("5\n");	

		
		hipFree(Noise_d);		
	}
	else if(incEFAC > 1 || incEQUAD > 1){

                 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
                 checkCudaError(err);
                 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
                 checkCudaError(err);

		
		//printf("here \n");
		MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, G,Noise_d,GlobalGmat_d,NG_d);



		status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, NG_d, N, beta, GG_d, G);
		checkStatus(status);


		status = culaDeviceDpotrf('L', G, GG_d, G);
		checkStatus(status);

		calc_det<<< 1, 1 >>>( GG_d, dettemp_d, G);
		err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
		checkCudaError(err);
		likeVals[0]=dettemp[0];

		status = culaDeviceDpotri('L', G, GG_d, G);
		checkStatus(status);

		status =  culaDeviceDsymm('R', 'L', N, G, alpha, GG_d, G, GlobalGmat_d, N, beta, NG_d, N);
		checkStatus(status);

		status =  culaDeviceDgemm('N', 'T', N, N, G, alpha, NG_d, N, GlobalGmat_d, N, beta, GNG_d, N);
		checkStatus(status);

		status = culaDeviceDgemv('N', N, N, alpha, GNG_d, N, resvec_d, 1, beta, GNGd_d, 1);
		checkStatus(status);
     

		err = hipMemcpy(GNGd, GNGd_d, sizeof(double)*N, hipMemcpyDeviceToHost);
		checkCudaError(err);
		likeVals[1]=0;
		for(int i =0;i < N; i++){likeVals[1] += resvec[i]*GNGd[i]; }
		
		make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,DMVec_d,N,RF,DMF, incRED, incDM);

		status =  culaDeviceDgemm('N', 'N', N, F, N, alpha, GNG_d, N, FMatrix_d, N, beta, NF_d, N);
		checkStatus(status);

		hipFree(Noise_d);	
	}


	 
  	 status =  culaDeviceDgemm('T', 'N', F, F, N, alpha, FMatrix_d, N, NF_d, N, beta, FNF_d, F);
	 checkStatus(status);
	
 	 status = culaDeviceDgemv('T', N, F, alpha, NF_d, N, resvec_d, 1, beta, NFd_d, 1);
     checkStatus(status);
	
	 err = hipMemcpy(FNFvec, FNF_d, sizeof(double)*F*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 	
	 err = hipMemcpy(NFd, NFd_d, sizeof(double)*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

		for(int f1=0;f1<F; f1++){
			for(int f2=0;f2<F; f2++){
 
				FNF[f2][f1]=FNFvec[f1*F + f2];
				//printf("GPUFNF: %i %i %g \n",f1,f2);
			}
		}

	hipFree(dettemp_d);
	hipFree(Freqs_d);
	hipFree(BATvec_d);
    	hipFree(FMatrix_d);
   	 hipFree(NF_d);
	hipFree(FNF_d);
	hipFree(resvec_d);
	hipFree(NFd_d);
	hipFree(DMVec_d);
	
	hipFree(NG_d);
   	hipFree(GG_d);
	hipFree(GNG_d);
	hipFree(GNGd_d);

	
	free(FNFvec);
	free(dettemp);
	free(GNGd);
	
}



extern "C" void LRedMarginNumGPUWrapper_( double *resvec, double *BATvec, double *Noise, double *likeVals, int N, int G){

	//printf("%i %i \n", SF,F);
	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	double *dettemp_d;

	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	
		
	double *NG_d;
	double *GG_d;
	double *GNG_d;
	double *GNGd_d;
	
	double *GNGd;
	GNGd = (double*)malloc(sizeof(double)*N);

	

	hipError_t err;
	culaStatus status;
    
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
      
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);

   	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GNGd_d, sizeof(double)*N );
	 checkCudaError(err);

	 
   	 err = hipMalloc( (void **)&NG_d, sizeof(double)*N*G);
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GG_d, sizeof(double)*G*G );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&GNG_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 
   // copy vectors from CPU to GPU
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;

	 MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, G,Noise_d,GlobalGmat_d,NG_d);
	 

 	 double alpha=1.0;
 	 double beta=0.0; 
 	 status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, NG_d, N, beta, GG_d, G);
	 checkStatus(status);


 	 status = culaDeviceDpotrf('L', G, GG_d, G);
     	checkStatus(status);

	calc_det<<< 1, 1 >>>( GG_d, dettemp_d, G);
	err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     	likeVals[0]=dettemp[0];
     
  	 status = culaDeviceDpotri('L', G, GG_d, G);
     	checkStatus(status);
     
  	 status =  culaDeviceDsymm('R', 'L', N, G, alpha, GG_d, G, GlobalGmat_d, N, beta, NG_d, N);
	 checkStatus(status);
	 
  	 status =  culaDeviceDgemm('N', 'T', N, N, G, alpha, NG_d, N, GlobalGmat_d, N, beta, GNG_d, N);
	 checkStatus(status);
	 
  	 status = culaDeviceDgemv('N', N, N, alpha, GNG_d, N, resvec_d, 1, beta, GNGd_d, 1);
     	checkStatus(status);
     

	 err = hipMemcpy(GNGd, GNGd_d, sizeof(double)*N, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 likeVals[1]=0;
  	 for(int i =0;i < N; i++){likeVals[1] += resvec[i]*GNGd[i];}


	hipFree(dettemp_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
	hipFree(resvec_d);
	
    hipFree(NG_d);
    hipFree(GG_d);
	hipFree(GNG_d);
	hipFree(GNGd_d);

	
	free(dettemp);
	free(GNGd);
	
}



extern "C" void vHRedDMMarginGPUWrapper_(double *Res, double *BatVec, double *NoiseVec, double *DMVec, double *SpecInfo, double *likeInfo, double *FactorialList, int N, int G)
{


	hipError_t err;
	culaStatus status;
	
	double secday=24*60*60;
	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecInfo[1];
	double dmmodelalpha=SpecInfo[3];
	
	double covconst=gsl_sf_gamma(1-modelalpha)*sin(0.5*M_PI*modelalpha);
	double dmcovconst=gsl_sf_gamma(1-dmmodelalpha)*sin(0.5*M_PI*dmmodelalpha);
	
	double gwamp=pow(10.0,SpecInfo[0]);
	double gwampsquared=gwamp*gwamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-modelalpha)))/(pow(flo,(modelalpha-1)));
	
	double dmamp=pow(10.0,SpecInfo[2]);
	double dmampsquared=dmamp*dmamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-dmmodelalpha)))/(pow(flo,(dmmodelalpha-1)));
	
	SpecInfo[0]=gwampsquared;
	SpecInfo[2]=dmampsquared;
	
	SpecInfo[4]=covconst;
	SpecInfo[5]=dmcovconst;
	
	double *Res_d;
	double *BatVec_d;
	double *NoiseVec_d;
	double *DMVec_d;
	double *SpecInfo_d;
	double *CovMatrix_d;

  
  	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&BatVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&NoiseVec_d, sizeof(double)*N );
	 checkCudaError(err);
 	 err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&SpecInfo_d, sizeof(double)*6 );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 

     err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( BatVec_d, BatVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( NoiseVec_d, NoiseVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
  	 err = hipMemcpy( DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy(SpecInfo_d, SpecInfo, sizeof(double)*6, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;//((G + dimBlock.x - 1) / dimBlock.x,(N + dimBlock.y - 1) / dimBlock.y);
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
	 MakeDMcov<<<dimGrid, dimBlock>>>(CovMatrix_d, BatVec_d, NoiseVec_d, DMVec_d, SpecInfo_d, N,N);
	 

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	
	double *GRes;
	GRes =(double*)malloc(sizeof(double)*G);
	double *WorkingGRes;
	WorkingGRes =(double*)malloc(sizeof(double)*G);
	

	// declare GPU copies

	double *CG_d;
	double *GCG_d;
	double *GRes_d;
	double *dettemp_d;


   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);

	 
	  	 err = hipMalloc( (void **)&CG_d, sizeof(double)*N*G );
	 checkCudaError(err);
	  	 err = hipMalloc( (void **)&GCG_d, sizeof(double)*G*G );
	 checkCudaError(err);
	  	 err = hipMalloc( (void **)&GRes_d, sizeof(double)*G );
	 checkCudaError(err);

   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	// err = hipMemcpy(CovMatrix_d, CovMatrix, sizeof(double)*N*N, hipMemcpyHostToDevice );
 	// checkCudaError(err);

 	 
	int carryOn=0;
  	double alpha=1.0;
 	double beta=0.0; 

 	status =  culaDeviceDsymm('L', 'U', N, G, alpha, CovMatrix_d, N, GlobalGmat_d, N, beta, CG_d, N);
	checkStatus(status);
	//printf("done first linalg\n");
  	status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, CG_d, N, beta, GCG_d, G);
	checkStatus(status);

	 status = culaDeviceDgemv('T', N, G, alpha, GlobalGmat_d, N, Res_d, 1, beta, GRes_d, 1);
     	checkStatus(status);
 	 err = hipMemcpy(GRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);	 
 	 //	printf("do chol\n");
		
 	 status = culaDeviceDpotrf('L', G, GCG_d, G);
     	checkStatusCarryOn(status,carryOn);

	calc_det<<< 1, 1 >>>( GCG_d, dettemp_d, G);
	err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
	checkCudaError(err);
     	likeInfo[0]=dettemp[0];
    // printf("det: %g \n",likeInfo[0]);
    
	if(carryOn == 1){
     	// printf("Bad chol\n");
     	 
		hipFree(dettemp_d);
		hipFree(Res_d);
		hipFree(BatVec_d);
		hipFree(NoiseVec_d);
		hipFree(SpecInfo_d);
		hipFree(CovMatrix_d);
		hipFree(CG_d);
		hipFree(GCG_d);
		hipFree(GRes_d);
		 hipFree(DMVec_d);
		free(dettemp);
		free(GRes);
		free(WorkingGRes);
		
		
		return;

  	 }

 	 status=culaDeviceDpotrs('L', G, 1, GCG_d, G, GRes_d, G);
	 checkStatus(status);
	 err = hipMemcpy(WorkingGRes, GRes_d, sizeof(double)*G, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

	 double sum=0;
	 for(int i=0; i<G;i++){sum=sum+GRes[i]*WorkingGRes[i]; }
	 likeInfo[1]=sum;
	 //printf("like: %g \n",sum);
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
  	 hipFree(BatVec_d);
 	 hipFree(NoiseVec_d);
 	 hipFree(SpecInfo_d);
	 hipFree(CovMatrix_d);
 	 hipFree(CG_d);
	 hipFree(GCG_d);
	 hipFree(GRes_d);
	 hipFree(DMVec_d);
  	 free(dettemp);
   	 free(GRes);
  	 free(WorkingGRes);
  


   return;
}


extern "C" void vHRedDMGPUWrapper_(double *SpecInfo, double *BatVec, double *Res, double *NoiseVec, double *DMVec, double *likeInfo, int N)
{

	hipError_t err;
	culaStatus status;

	double secday=24*60*60;
	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecInfo[1];
	double dmmodelalpha=SpecInfo[3];
	
	double covconst=gsl_sf_gamma(1-modelalpha)*sin(0.5*M_PI*modelalpha);
	double dmcovconst=gsl_sf_gamma(1-dmmodelalpha)*sin(0.5*M_PI*dmmodelalpha);
	
	double gwamp=pow(10.0,SpecInfo[0]);
	double gwampsquared=gwamp*gwamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-modelalpha)))/(pow(flo,(modelalpha-1)));
	
	double dmamp=pow(10.0,SpecInfo[2]);
	double dmampsquared=dmamp*dmamp*(pow((365.25*secday),2)/(12*M_PI*M_PI))*(pow(365.25,(1-dmmodelalpha)))/(pow(flo,(dmmodelalpha-1)));
	
	SpecInfo[0]=gwampsquared;
	SpecInfo[2]=dmampsquared;
	
	SpecInfo[4]=covconst;
	SpecInfo[5]=dmcovconst;
	
	double *Res_d;
	double *BatVec_d;
	double *NoiseVec_d;
	double *DMVec_d;
	double *SpecInfo_d;
	double *CovMatrix_d;

  
  	 err = hipMalloc( (void **)&Res_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&BatVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&NoiseVec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&SpecInfo_d, sizeof(double)*6 );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&CovMatrix_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 

     	 err = hipMemcpy( Res_d, Res, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( BatVec_d, BatVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( NoiseVec_d, NoiseVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
   	 err = hipMemcpy( DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy(SpecInfo_d, SpecInfo, sizeof(double)*6, hipMemcpyHostToDevice );
 	 checkCudaError(err);

 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;//((G + dimBlock.x - 1) / dimBlock.x,(N + dimBlock.y - 1) / dimBlock.y);
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 
	 MakeDMcov<<<dimGrid, dimBlock>>>(CovMatrix_d, BatVec_d, NoiseVec_d, DMVec_d, SpecInfo_d, N,N);
	 

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	

	double *WorkingRes;
	WorkingRes = (double*)malloc(sizeof(double)*N);

	// declare GPU copies
	double *dettemp_d;



   // Allocate memory on GPU
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);

   // copy vectors from CPU to GPU
   	 err = hipMemcpy( dettemp_d, dettemp, sizeof(double), hipMemcpyHostToDevice );
 	 checkCudaError(err);

	 int carryOn=0;
 	 status = culaDeviceDpotrf('L', N, CovMatrix_d, N);
     	 checkStatusCarryOn(status,carryOn);
     


	 calc_det<<< 1, 1 >>>( CovMatrix_d, dettemp_d, N);
	 err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
  	 checkCudaError(err);
     	 likeInfo[0]=dettemp[0];
     
          if(carryOn == 1){

     	 	 hipFree(dettemp_d);
	 	 hipFree(Res_d);
		 hipFree(BatVec_d);
		 hipFree(NoiseVec_d);
		 hipFree(SpecInfo_d);
		 hipFree(CovMatrix_d);
		 hipFree(DMVec_d);
	  	 free(dettemp);
	  	 free(WorkingRes);
	  	 return;
  	 }
	

 	 status=culaDeviceDpotrs('L', N, 1, CovMatrix_d, N, Res_d, N);
	 checkStatus(status);
	 err = hipMemcpy(WorkingRes, Res_d, sizeof(double)*N, hipMemcpyDeviceToHost);

  	 checkCudaError(err);

	 double sum=0;
	 	
	for(int i=0; i<N;i++){
		sum=sum+Res[i]*WorkingRes[i];
	}

	 likeInfo[1]=sum;
	 
	 hipFree(dettemp_d);
 	 hipFree(Res_d);
  	 hipFree(BatVec_d);
 	 hipFree(NoiseVec_d);
 	 hipFree(SpecInfo_d);
	 hipFree(CovMatrix_d);
 	hipFree(DMVec_d);
  	 free(dettemp);
  	 free(WorkingRes);
  	 
  


   return;
}

extern "C" void LRedDMMarginGPUWrapper_(double *Freqs, double *resvec, double *BATvec, double *DMVec, double *Noise, double **FNF, double *NFd, double *likeVals, int N, int F, int G){

	double *dettemp;
	dettemp = (double*)malloc(sizeof(double));
	dettemp[0]=0;
	
	double *dettemp_d;

	double *Freqs_d;
	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	double *DMVec_d;
	
		
	double *NG_d;
	double *GG_d;
	double *GNG_d;
	double *GNGd_d;
	
	double *GNGd;
	GNGd = (double*)malloc(sizeof(double)*N);

	
	double *FMatrix_d;
	double *NF_d;	
	double *FNF_d;
	double *NFd_d;
	
	double *FNFvec;
	FNFvec = (double*)malloc(sizeof(double)*F*F);

	hipError_t err;
	culaStatus status;
    
  	 err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	 checkCudaError(err);
      
  	 err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GNGd_d, sizeof(double)*N );
	 checkCudaError(err);

	 
   	 err = hipMalloc( (void **)&NG_d, sizeof(double)*N*G);
	 checkCudaError(err);
	 err = hipMalloc( (void **)&GG_d, sizeof(double)*G*G );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&GNG_d, sizeof(double)*N*N );
	 checkCudaError(err);
	 
   	 err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&NF_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&FNF_d, sizeof(double)*F*F );
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&NFd_d, sizeof(double)*F);
	 checkCudaError(err);	
	 
   // copy vectors from CPU to GPU
   	 err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;
	 dimGrid.x=(N + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;

	 MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, G,Noise_d,GlobalGmat_d,NG_d);
	 

 	 double alpha=1.0;
 	 double beta=0.0; 
 	 status =  culaDeviceDgemm('T', 'N', G, G, N, alpha, GlobalGmat_d, N, NG_d, N, beta, GG_d, G);
	 checkStatus(status);


 	 status = culaDeviceDpotrf('L', G, GG_d, G);
 	checkStatus(status);

	calc_det<<< 1, 1 >>>( GG_d, dettemp_d, G);
	err = hipMemcpy( dettemp, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
 	checkCudaError(err);
 	likeVals[0]=dettemp[0];
     
  	 status = culaDeviceDpotri('L', G, GG_d, G);
 	 checkStatus(status);
     
  	 status =  culaDeviceDsymm('R', 'L', N, G, alpha, GG_d, G, GlobalGmat_d, N, beta, NG_d, N);
	 checkStatus(status);
	 
  	 status =  culaDeviceDgemm('N', 'T', N, N, G, alpha, NG_d, N, GlobalGmat_d, N, beta, GNG_d, N);
	 checkStatus(status);
	 
  	 status = culaDeviceDgemv('N', N, N, alpha, GNG_d, N, resvec_d, 1, beta, GNGd_d, 1);
     	checkStatus(status);
     

	 err = hipMemcpy(GNGd, GNGd_d, sizeof(double)*N, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 likeVals[1]=0;
  	 for(int i =0;i < N; i++){likeVals[1] += resvec[i]*GNGd[i];}

 	 make_DMfmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,DMVec_d, N,F);
 	 
   	 status =  culaDeviceDgemm('N', 'N', N, F, N, alpha, GNG_d, N, FMatrix_d, N, beta, NF_d, N);
	 checkStatus(status);
	 
  	 status =  culaDeviceDgemm('T', 'N', F, F, N, alpha, FMatrix_d, N, NF_d, N, beta, FNF_d, F);
	 checkStatus(status);
	
 	 status = culaDeviceDgemv('T', N, F, alpha, NF_d, N, resvec_d, 1, beta, NFd_d, 1);
     checkStatus(status);
	
	 err = hipMemcpy(FNFvec, FNF_d, sizeof(double)*F*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 	
	 err = hipMemcpy(NFd, NFd_d, sizeof(double)*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

		for(int f1=0;f1<F; f1++){
			for(int f2=0;f2<F; f2++){
 
				FNF[f2][f1]=FNFvec[f1*F + f2];
				//printf("GPUFNF: %i %i %g \n",f1,f2);
			}
		}

	hipFree(dettemp_d);
	hipFree(Freqs_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
    hipFree(FMatrix_d);
    hipFree(NF_d);
	hipFree(FNF_d);
	hipFree(resvec_d);
	hipFree(NFd_d);
	
    hipFree(NG_d);
    hipFree(GG_d);
	hipFree(GNG_d);
	hipFree(GNGd_d);

	
	free(FNFvec);
	free(dettemp);
	free(GNGd);
	
}





 	 

 extern "C" void copy_floatgmat_(float *G, int N){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalGmatFloat_d, sizeof(float)*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalGmatFloat_d, G, sizeof(float)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}


extern "C" void copy_gmat_(double *G, int N){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalGmat_d, sizeof(double)*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalGmat_d, G, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}

extern "C" void copy_staticgmat_(double *G, int M, int N){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalStaticGmat_d, sizeof(double)*N*M );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalStaticGmat_d, G, sizeof(double)*N*M, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}

extern "C" void copy_staticumat_(double *G, int M, int N){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalStaticUGmat_d, sizeof(double)*M*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalStaticUGmat_d, G, sizeof(double)*M*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}


