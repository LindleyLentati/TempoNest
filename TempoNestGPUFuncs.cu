#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <gsl/gsl_sf_gamma.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "dgesvd.h"
#include <cula_lapack_device.h>
#include <cula_blas_device.h>


#define BLOCK_SIZE 16

//double *GlobalGmat_d;
//double *GlobalStaticGmat_d;
//double *GlobalStaticUGmat_d;
//float *GlobalGmatFloat_d;
//double *GlobalStaticDmat_d;
double *GlobalTotalMatrix_d;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)

double iter_factorialGPU(unsigned int n)
{
    double ret = 1;
    for(unsigned int i = 1; i <= n; ++i)
        ret *= i;
    return ret;
}


void checkStatus(culaStatus status)
{
    char buf[256];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    culaShutdown();
    exit(EXIT_FAILURE);
}

void checkStatusCarryOn(culaStatus status, int &CarryOn)
{
    char buf[256];

    if(!status){
    	CarryOn=0;
        return;
        }

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    //printf("%s\n", buf);
    CarryOn=1;
    
    culaShutdown();
    culaStatus status2;
	status2 = culaInitialize();
	
}


void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    culaShutdown();
    exit(EXIT_FAILURE);
}


// simple kernel function that adds two vectors
__global__ void vect_add(double *a, double *b, int N)
{
   int Bidx = blockIdx.x;
   //a[Bidx*N+Tidx] = a[Bidx*N+Tidx] + b[Bidx*N+Tidx]; 
   for(int i =0; i < N; i++){
   		a[Bidx*N+i] = a[Bidx*N+i] + b[Bidx*N+i]; 
   		}
}


// simple kernel function that calcs det of a matrix
__global__ void calc_det(double *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void Floatcalc_det(float *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void calc_detFloat(float *a, double *det, int N)
{
	
	det[0]=0;
   for(int i =0; i < N; i++){
   		det[0]+=log(a[i*N+i]);
   		}
   		det[0]=det[0]*2;
   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void calc_DiagLike(double *Vec, double *Noise, int N, double *val)
{
	
	val[0]=0;
	for(int i =0; i < N; i++){
   		val[0]+=Vec[i]*Vec[i]*Noise[i];
	}

   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void calc_DotLike(double *Vec1, double *Vec2, int N, double *val)
{
	
	val[0]=0;
	for(int i =0; i < N; i++){
   		val[0]+=Vec1[i]*Vec2[i];
   		//printf("GPU copy %i %g %g\n", i,Vec1[i],Vec2[i]);
	}

   		
   		
}

// simple kernel function that calcs det of a matrix
__global__ void copyvec(double *Vec1, double *Vec2, int N)
{
	
		int Bidx = blockIdx.x;
		 __syncthreads();
   		Vec1[Bidx]=Vec2[Bidx];
   		//printf("copy: %i %g\n",Bidx, Vec1[Bidx]);
   		
}

/*
__global__ void Makecov(double *A_d, double *BATvec, double *NoiseVec, double *SpecParm, int Aheight, int Awidth) {

	// Each thread computes one element of C
	// by accumulating results into Cvalue
	

	double LongestPeriod=1.0/pow(10.0,-5); //
	double flo=1.0/LongestPeriod;

	double modelalpha=SpecParm[1];
	double gwampsquared=SpecParm[0];
	double covconst=SpecParm[2];
	

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;
	double timdiff= BATvec[row] - BATvec[col];	
	double tau=2.0*M_PI*fabs(timdiff);
	double covsum=0;

	for(int k=0; k <=10; k++){
	
		    double ret = 1;
			for(unsigned int i = 1; i <= 2*k; ++i){
				ret *= (double)i;
			//	printf("Ret: %i %g \n",i,ret);
			}
    
			covsum=covsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(ret*(2*k+1-modelalpha));
			//printf("covsum: %i %i %i %g \n",row,col,k,covsum);

	}

	A_d[row * Awidth + col]=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);

	if(row==col){
		A_d[row * Awidth + col] += NoiseVec[row];
	}

}


__global__ void MakeDMcov(double *A_d, double *BATvec, double *NoiseVec, double *DMVec, double *SpecParm, int Aheight, int Awidth) {

	// Each thread computes one element of C
	// by accumulating results into Cvalue
	

	double LongestPeriod=1.0/pow(10.0,-5);
	double flo=1.0/LongestPeriod;

	double gwampsquared=SpecParm[0];
	double modelalpha=SpecParm[1];
	double covconst=SpecParm[2];
	
	double dmampsquared=SpecParm[3];
	double dmmodelalpha=SpecParm[4];
	double dmcovconst=SpecParm[5];
	

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;
	double timdiff= BATvec[row] - BATvec[col];	
	double tau=2.0*M_PI*fabs(timdiff);
	double covsum=0;
	double dmcovsum=0;

	for(int k=0; k <=5; k++){
	
		    double ret = 1;
			for(unsigned int i = 1; i <= 2*k; ++i){
				ret *= (double)i;
			}
    
			covsum=covsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(ret*(2*k+1-modelalpha));
			
			dmcovsum=dmcovsum+pow(-1.0,k)*(pow(flo*tau,2*k))/(ret*(2*k+1-dmmodelalpha));

	}
	
	double gwpart=0;
	if(SpecParm[0] !=0 )gwpart=gwampsquared*(covconst*pow((flo*tau),(modelalpha-1)) - covsum);
	
	double dmpart=0;
	if(SpecParm[3] !=0 )dmpart=dmampsquared*(dmcovconst*pow((flo*tau),(dmmodelalpha-1)) - dmcovsum)*DMVec[row]*DMVec[col];

	A_d[row * Awidth + col]= gwpart+dmpart;

	if(row==col){
		A_d[row * Awidth + col] += NoiseVec[row];
	}

	//printf("%i %i %g\n",row,col,A_d[row * Awidth + col]);

}

*/
__global__ void MatMulKernel(int Arow,int Acol,int Brow, int Bcol,double *A,double *B,double *C)
{

	int Crow=Arow;
	int Ccol=Bcol;
    double Ctemp = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    __syncthreads();

	if(row < Arow && col < Bcol) {
		//if(row<32)printf("GPUNT: %i %i %g %g \n", row, col, B[col * Brow + row], A[row] );
   		Ctemp = A[row] * B[col * Brow + row];
						  //GGTest[col*N + row]


	   C[col*Crow+row] = Ctemp;
	}
	  // 
}


__global__ void SimpleDiagMatMulKernel(int N,int T,double *Noise_d,double *TMatrix_d,double *NTMatrix_d)
{

	
    
    __syncthreads();


		for(int i=0;i<T; i++){
			for(int j=0;j<N; j++){
				//if(i ==28)printf("GPU SDMMK %i %i %g %g \n",i,j,TMatrix_d[i*N + j],Noise_d[j]);
				NTMatrix_d[i*N + j]=TMatrix_d[i*N + j]*Noise_d[j];
			}
		}		
}


/*
extern "C" void WhiteMarginGPUWrapper_(void *context, double *TNDMVec, double *resvec, double *Noise, int N, int D, int NTime, int NJumps, double *likevals){



	double *resvec_d;
	double *Noise_d;


	double *DMatrix_d;
	double *NT_d;	
	double *TNT_d;
	double *NTd_d;


	hipError_t err;
	culaStatus status;

	// allocate memory on GPU
	err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	checkCudaError(err);
	err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	checkCudaError(err);



	err = hipMalloc( (void **)&DMatrix_d, sizeof(double)*N*D );
	checkCudaError(err);
	err = hipMalloc( (void **)&NT_d, sizeof(double)*N*D );
	checkCudaError(err);
	err = hipMalloc( (void **)&TNT_d, sizeof(double)*D*D );
	checkCudaError(err);	 
	err = hipMalloc( (void **)&NTd_d, sizeof(double)*D);
	checkCudaError(err);	

	// copy vectors from CPU to GPU
	err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	checkCudaError(err);
	err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
	checkCudaError(err);

 	 

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid;
	
/////////////////////////////////////////////////////////////////////////////////////////////  
/////////////////////////Form the Design Matrix////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////// 	


	if(D != NTime+NJumps){
	
		err = hipMemcpy( DMatrix_d, TNDMVec, sizeof(double)*D*N, hipMemcpyHostToDevice );
		checkCudaError(err);
	
		double *U_d;
		double *V_d;
		double *S_d;
	
		err = hipMalloc( (void **)&U_d, sizeof(double)*N*N );
		checkCudaError(err);
		err = hipMalloc( (void **)&V_d, sizeof(double)*D*D );
		checkCudaError(err);
		err = hipMalloc( (void **)&S_d, sizeof(double)*D );
		checkCudaError(err);
	
	
		culaDeviceDgesvd('O','N', N, D, DMatrix_d, N, S_d, U_d, N, V_d, D);
	
		hipFree(V_d);
		hipFree(S_d);
		hipFree(U_d);
		
		hipDeviceSynchronize();	
	}
	else{
		DMatrix_d=GlobalStaticDmat_d;
		hipDeviceSynchronize();	
	}
	


///////////////////////////////////////////////////////////////////////////////////////////  
/////////////////////////Do the Algebra///////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////// 	
 	 
 	 
	//printf("entered 5 %i %i\n",T,N);
	
	
	dimGrid.x=(D + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;	

	MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, D,Noise_d,DMatrix_d,NT_d);
	//SimpleDiagMatMulKernel<<<1,1>>>(N, T, Noise_d, TMatrix_d, NT_d);
	hipDeviceSynchronize();

	double alpha=1.0;
	double beta=0.0; 
	

	status =  culaDeviceDgemm('T', 'N', D, D, N, alpha, DMatrix_d, N, NT_d, N, beta, TNT_d, D);
	checkStatus(status);

	hipDeviceSynchronize();
	

	status = culaDeviceDgemv('T', N, D, alpha, NT_d, N, resvec_d, 1, beta, NTd_d, 1);
	checkStatus(status);
	
	hipDeviceSynchronize();
	

	
	//printf("entered 6: %i \n", T);
	double *dettemp_d;
	double *tempval=new double[1];
	err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	checkCudaError(err);

	int carryOn=0;
	status = culaDeviceDpotrf('L', D, TNT_d, D);
	hipDeviceSynchronize();
	//printf("entered 6.25\n");
	
	checkStatusCarryOn(status,carryOn);
	
	hipDeviceSynchronize();

	//printf("entered 6.5\n");
	calc_det<<< 1, 1 >>>( TNT_d, dettemp_d, D);
	err = hipMemcpy( tempval, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
	checkCudaError(err);
	likevals[0]=tempval[0];
	
	hipDeviceSynchronize();
     
	if(carryOn == 1){


		hipFree(resvec_d);
		hipFree(Noise_d);



		hipFree(DMatrix_d);
		hipFree(NT_d);
		hipFree(TNT_d);
		hipFree(NTd_d);
		
		hipFree(dettemp_d);
		delete(tempval);
		return;
	}

	//printf("entered 7\n");
	double *WorkVec_d;
	err = hipMalloc( (void **)&WorkVec_d, sizeof(double)*D );
	checkCudaError(err);
	copyvec<<< D, 1 >>>(WorkVec_d, NTd_d, D);
	
	hipDeviceSynchronize();
	

	status=culaDeviceDpotrs('L', D, 1, TNT_d, D, WorkVec_d, D);
	checkStatus(status);
	
	hipDeviceSynchronize();
	
	double *freqlike_d;
	err = hipMalloc( (void **)&freqlike_d, sizeof(double));
	checkCudaError(err);

	calc_DotLike<<< 1, 1 >>>(WorkVec_d, NTd_d, D, freqlike_d);
	
	hipDeviceSynchronize();
	err = hipMemcpy( tempval, freqlike_d, sizeof(double), hipMemcpyDeviceToHost);
	checkCudaError(err);
	likevals[1]=tempval[0];

	


	hipFree(resvec_d);
	hipFree(Noise_d);

	hipFree(DMatrix_d);
	hipFree(NT_d);
	hipFree(TNT_d);
	hipFree(NTd_d);
	
	hipFree(WorkVec_d);
	hipFree(dettemp_d);
	delete(tempval);

}
*/


// simple kernel function that calculates the FMatrix
__global__ void make_fmatrix(double *TMatrix_d,double *Freqs_d, double *ObsFreqs_d, double *BATvec_d, double *DMVec_d, int *SysGroups_d, int *BandFreqs, int N,int RF, int DMF, int BandNoiseCoeff, int GroupNoiseCoeff,  int incRED, int incDM, int incBandNoise, int incGroupNoise, int ReplaceTMatrix, int TimetoMargin, int numTime, int numJumps, double *DMatrix_d)
{

	int Bidx = blockIdx.x;


	if(TimetoMargin != numJumps + numTime){
		for(int i=0;i<TimetoMargin;i++){
			TMatrix_d[i*N + Bidx]=DMatrix_d[i*N + Bidx];
		}
	}

	int startpos=0;
	if(incRED !=0){
		if(ReplaceTMatrix==0){
			for(int i=0;i<RF/2;i++){
				TMatrix_d[(TimetoMargin+i)*N + Bidx]=cos(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
				TMatrix_d[(TimetoMargin+i+RF/2)*N + Bidx]=sin(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
			}
		}
		startpos=RF;
	}

	
      if(incDM !=0){
		if(ReplaceTMatrix==0){
		        for(int i=0;i<DMF/2;i++){
				//if(Bidx==0)printf("D: %i %i %g %g \n", Bidx,i,1.0/Freqs_d[i], DMVec_d[Bidx]);
		                TMatrix_d[(TimetoMargin+startpos+i)*N + Bidx]=cos(2*M_PI*Freqs_d[startpos+i]*BATvec_d[Bidx])*DMVec_d[Bidx];
		                TMatrix_d[(TimetoMargin+startpos+i+DMF/2)*N + Bidx]=sin(2*M_PI*Freqs_d[startpos+i]*BATvec_d[Bidx])*DMVec_d[Bidx];
		        }
		}
		startpos=startpos+DMF;
       }

	if(incBandNoise > 0){

		
		for(int b = 0; b < incBandNoise; b++){	

			if(ReplaceTMatrix==0){
				int startfreq = BandFreqs[b*3+0];
				int stopfreq = BandFreqs[b*3+1];
				int BandScale = BandFreqs[b*3+2];


				for(int i=0;i<BandNoiseCoeff/2;i++){
					if(ObsFreqs_d[Bidx] > startfreq && ObsFreqs_d[Bidx] < stopfreq){
						TMatrix_d[(TimetoMargin+startpos+i)*N + Bidx]=cos(2*M_PI*Freqs_d[startpos+i]*BATvec_d[Bidx]);
						TMatrix_d[(TimetoMargin+startpos+i+BandNoiseCoeff/2)*N + Bidx]=sin(2*M_PI*Freqs_d[startpos+i]*BATvec_d[Bidx]);
					}
					else{
			
						TMatrix_d[(TimetoMargin+startpos+i)*N + Bidx]=0;
						TMatrix_d[(TimetoMargin+startpos+i+BandNoiseCoeff/2)*N + Bidx]=0;
		
					}
				}
			}
			startpos=startpos+BandNoiseCoeff;

		}


       }

	//printf("In GPU : %i \n", incGroupNoise); 
	if(incGroupNoise > 0){
		for(int g = 0; g < incGroupNoise; g++){

			for(int i=0;i<GroupNoiseCoeff/2;i++){
				//printf("GPU Groups %i %i \n", Bidx, SysGroups_d[Bidx]);
				if(SysGroups_d[Bidx] == g+1){
					TMatrix_d[(TimetoMargin+startpos+i)*N + Bidx]=cos(2*M_PI*Freqs_d[startpos+i]*BATvec_d[Bidx]);
					TMatrix_d[(TimetoMargin+startpos+i+GroupNoiseCoeff/2)*N + Bidx]=sin(2*M_PI*Freqs_d[startpos+i]*BATvec_d[Bidx]);
				}
				else{
		
					TMatrix_d[(TimetoMargin+startpos+i)*N + Bidx]=0;
					TMatrix_d[(TimetoMargin+startpos+i+GroupNoiseCoeff/2)*N + Bidx]=0;
	
				}
			}



			startpos=startpos+GroupNoiseCoeff;

		}

	}
}
/*


// simple kernel function that calculates the FMatrix
__global__ void add_EcorrToFMatrix(double *FMatrix_d, double *EMatrix_d, int Nobs, int FSize, int EpochSize)
{

	int Bidx = blockIdx.x;
        for(int i=0;i<EpochSize;i++){
                FMatrix_d[(FSize+i)*Nobs + Bidx]= EMatrix_d[i*Nobs + Bidx];
        }


}
*/
/*

// simple kernel function that calculates the FMatrix
__global__ void make_DMfmatrix(double *FMatrix_d,double *Freqs_d, double *BATvec_d, double *DMVec_d, int N,int F)
{

	int Bidx = blockIdx.x;
	
	for(int i=0;i<F/4;i++){
		
			FMatrix_d[i*N + Bidx]=cos(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
			FMatrix_d[(i+F/4)*N  + Bidx]=cos(2*M_PI*Freqs_d[i]*BATvec_d[Bidx])*DMVec_d[Bidx];
			FMatrix_d[(i+F/2)*N + Bidx]=sin(2*M_PI*Freqs_d[i]*BATvec_d[Bidx]);
			FMatrix_d[(i+3*F/4)*N + Bidx]=sin(2*M_PI*Freqs_d[i]*BATvec_d[Bidx])*DMVec_d[Bidx];
	}


}


__global__ void fastmake_fmatrix(double *FMatrix_d,double *Freqs_d, double *BATvec_d, int Aheight,int Awidth) {

	// Each thread computes one element of F
	// by accumulating results into Cvalue


	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	 __syncthreads();
	if(row >= Aheight || col >= Awidth) return;

	FMatrix_d[row * Awidth + col]=cos(2*M_PI*Freqs_d[col]*BATvec_d[row]);
	FMatrix_d[row*Awidth + col + Awidth/2]=sin(2*M_PI*Freqs_d[col]*BATvec_d[row]);

}
*/

/*

extern "C" void LRedGPUWrapper_(double *Freqs, double *resvec, double *BATvec, double *DMVec, double *Noise, double **FNF, double *NFd, int N, int RF,int DMF, int F, int incRED, int incDM){

	double *Freqs_d;
	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	double *DMVec_d;
	
	double *FMatrix_d;
	double *NF_d;	
	double *FNF_d;
	double *NFd_d;
	
	double *FNFvec;
	FNFvec = (double*)malloc(sizeof(double)*F*F);

	hipError_t err;
	culaStatus status;
      
  	 err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	 checkCudaError(err);
	 err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	 checkCudaError(err);
  	 err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	 checkCudaError(err);
     err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
     checkCudaError(err);

	 
   	 err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&NF_d, sizeof(double)*N*F );
	 checkCudaError(err);
   	 err = hipMalloc( (void **)&FNF_d, sizeof(double)*F*F );
	 checkCudaError(err);	 
   	 err = hipMalloc( (void **)&NFd_d, sizeof(double)*F);
	 checkCudaError(err);	
	 
         // copy vectors from CPU to GPU
   	 err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
 	 checkCudaError(err);
	 err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	 checkCudaError(err);
   	 err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
 	 err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);
     err = hipMemcpy( DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
     checkCudaError(err);

 	 
// 	 make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,N,F);
  	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	 dim3 dimGrid;

	 dimGrid.x=(F + dimBlock.x - 1)/dimBlock.x;
	 dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;
	 //fastmake_fmatrix<<<dimGrid, dimBlock>>>(FMatrix_d,Freqs_d,BATvec_d,N,F);
 	 //make_fmatrix<<< N, 1 >>>(FMatrix_d,Freqs_d,BATvec_d,DMVec_d,N,RF,DMF, 0, incRED, incDM, 0);

	 MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, F,Noise_d,FMatrix_d,NF_d);

 	 
	double alpha=1.0;
	double beta=0.0; 
	status =  culaDeviceDgemm('T', 'N', F, F, N, alpha, FMatrix_d, N, NF_d, N, beta, FNF_d, F);
	checkStatus(status);
	
 	 status = culaDeviceDgemv('T', N, F, alpha, NF_d, N, resvec_d, 1, beta, NFd_d, 1);
	 checkStatus(status);
	
	 err = hipMemcpy(FNFvec, FNF_d, sizeof(double)*F*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);
  	 	
	 err = hipMemcpy(NFd, NFd_d, sizeof(double)*F, hipMemcpyDeviceToHost);
  	 checkCudaError(err);

	for(int f1=0;f1<F; f1++){
		for(int f2=0;f2<F; f2++){

			FNF[f2][f1]=FNFvec[f1*F + f2];
		}
	}

	hipFree(Freqs_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
    	hipFree(FMatrix_d);
    	hipFree(NF_d);
	hipFree(FNF_d);
	hipFree(resvec_d);
	hipFree(NFd_d);
	hipFree(DMVec_d);
	free(FNFvec);
		
}
*/
/*
// simple kernel function that calculates the TMatrix
__global__ void make_Tmatrix(double *TMatrix_d, double *DMatrix_d, double *FMatrix_d, int N, int T, int D, int F)
{


 //   int row = blockIdx.y * blockDim.y + threadIdx.y;
  //  int col = blockIdx.x * blockDim.x + threadIdx.x;
     __syncthreads();
     
 //	int Bidx = blockIdx.x;
    int Bidx = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

//	for(int i=0;i<T;i++){
		if(i<D){
			TMatrix_d[i*N + Bidx]=DMatrix_d[i*N + Bidx];
		}
		else{
			if(i==D)printf("i==D %i
			TMatrix_d[i*N + Bidx]=FMatrix_d[(i-D)*N + Bidx];
		}
//	}
	
	
	
	 	int Bidx = blockIdx.x;


	for(int i=0;i<T;i++){
		if(i<D){
			TMatrix_d[i*N + Bidx]=DMatrix_d[i*N + Bidx];
		}
		else{
			//if(i==D)printf("i==D %i %g \n", Bidx, FMatrix_d[(i-D)*N + Bidx]);
			TMatrix_d[i*N + Bidx]=FMatrix_d[(i-D)*N + Bidx];
		}
	}

}

*/
// simple kernel function that adds powercoeff to TNT
__global__ void addCoeffsKernel(int T, int D,int F,double *TNT_d, double *powercoeffs_d)
{


     __syncthreads();
    for(int i =0; i < F; i++){
    	
    	TNT_d[(D+i)*T+D+i]+=1.0/powercoeffs_d[i];
    }


}


extern "C" void NewLRedMarginGPUWrapper_(void *context, double *TNDMVec, double *Freqs, double *ObsFreqs, double *powercoeff, double *resvec, double *BATvec, double *DMVec, double *Noise, int *SysGroups, int N, int RF,int DMF, int BandNoiseCoeff, int GroupNoiseCoeff, int D, int F, int T, int incRED, int incDM, int incBandNoise, int incGroupNoise, int NTime, int NJumps, double *likevals, int incNGJitter, int numNGJitterEpochs, int *BandInfo, int ReplaceTMatrix){


	//printf("entered 1\n");
	double *Freqs_d;
	double *ObsFreqs_d;
	double *powercoeff_d;
	double *resvec_d;
	double *BATvec_d;
	double *Noise_d;
	double *DMVec_d;
	int *SysGroups_d;
	int *BandInfo_d;
	
//	double *FMatrix_d;
	double *DMatrix_d;
//	double *TMatrix_d;
	double *NT_d;	
	double *TNT_d;
	double *NTd_d;


	hipError_t err;
	culaStatus status;

	// allocate memory on GPU
	err = hipMalloc( (void **)&Freqs_d, sizeof(double)*F );
	checkCudaError(err);
        err = hipMalloc( (void **)&ObsFreqs_d, sizeof(double)*N );
        checkCudaError(err);
	err = hipMalloc( (void **)&powercoeff_d, sizeof(double)*F );
	checkCudaError(err);
	err = hipMalloc( (void **)&resvec_d, sizeof(double)*N );
	checkCudaError(err);
	err = hipMalloc( (void **)&BATvec_d, sizeof(double)*N );
	checkCudaError(err);
	err = hipMalloc( (void **)&Noise_d, sizeof(double)*N );
	checkCudaError(err);
	err = hipMalloc( (void **)&DMVec_d, sizeof(double)*N );
	checkCudaError(err);
	err = hipMalloc( (void **)&SysGroups_d, sizeof(int)*N );
	checkCudaError(err);
	err = hipMalloc( (void **)&BandInfo_d, sizeof(int)*3*incBandNoise);
	checkCudaError(err);


//	err = hipMalloc( (void **)&FMatrix_d, sizeof(double)*N*F );
//	checkCudaError(err);

//	err = hipMalloc( (void **)&TMatrix_d, sizeof(double)*N*T );
//	checkCudaError(err);


	err = hipMalloc( (void **)&NT_d, sizeof(double)*N*T );
	checkCudaError(err);
	err = hipMalloc( (void **)&TNT_d, sizeof(double)*T*T );
	checkCudaError(err);	 
	err = hipMalloc( (void **)&NTd_d, sizeof(double)*T);
	checkCudaError(err);	

	// copy vectors from CPU to GPU
	err = hipMemcpy( Freqs_d, Freqs, sizeof(double)*F, hipMemcpyHostToDevice );
	checkCudaError(err);
        err = hipMemcpy( ObsFreqs_d, ObsFreqs, sizeof(double)*N, hipMemcpyHostToDevice );
        checkCudaError(err);
	err = hipMemcpy( powercoeff_d, powercoeff, sizeof(double)*F, hipMemcpyHostToDevice );
	checkCudaError(err);
	err = hipMemcpy(resvec_d, resvec, sizeof(double)*N, hipMemcpyHostToDevice );
	checkCudaError(err);
	err = hipMemcpy(BATvec_d, BATvec, sizeof(double)*N, hipMemcpyHostToDevice );
	checkCudaError(err);
	err = hipMemcpy( Noise_d, Noise, sizeof(double)*N, hipMemcpyHostToDevice );
	checkCudaError(err);
	err = hipMemcpy( DMVec_d, DMVec, sizeof(double)*N, hipMemcpyHostToDevice );
	checkCudaError(err);
	err = hipMemcpy( SysGroups_d, SysGroups, sizeof(int)*N, hipMemcpyHostToDevice );
	checkCudaError(err);
 	err = hipMemcpy( BandInfo_d, BandInfo, sizeof(int)*3*incBandNoise, hipMemcpyHostToDevice );
	checkCudaError(err);	 

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid;
	
/////////////////////////////////////////////////////////////////////////////////////////////  
/////////////////////////Form the Design Matrix////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////// 	

	//printf("entered 2\n");
	if(D != NTime+NJumps){

		err = hipMalloc( (void **)&DMatrix_d, sizeof(double)*N*D );
		checkCudaError(err);
		err = hipMemcpy( DMatrix_d, TNDMVec, sizeof(double)*D*N, hipMemcpyHostToDevice );
		checkCudaError(err);
	
		double *U_d;
		double *V_d;
		double *S_d;
	
		err = hipMalloc( (void **)&U_d, sizeof(double)*N*N );
		checkCudaError(err);
		err = hipMalloc( (void **)&V_d, sizeof(double)*D*D );
		checkCudaError(err);
		err = hipMalloc( (void **)&S_d, sizeof(double)*D );
		checkCudaError(err);
	
	
		culaDeviceDgesvd('O','N', N, D, DMatrix_d, N, S_d, U_d, N, V_d, D);
	
		hipFree(V_d);
		hipFree(S_d);
		hipFree(U_d);
		
		hipDeviceSynchronize();	
	}
	
   

///////////////////////////////////////////////////////////////////////////////////////////  
/////////////////////////Form the F Matrix/////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////// 	
	
	//printf("entered 3\n");
	make_fmatrix<<< N, 1 >>>(GlobalTotalMatrix_d,Freqs_d,ObsFreqs_d, BATvec_d,DMVec_d,SysGroups_d, BandInfo_d, N,RF,DMF, BandNoiseCoeff, GroupNoiseCoeff, incRED, incDM, incBandNoise, incGroupNoise, 				ReplaceTMatrix, D, NTime,NJumps,DMatrix_d);
 	 
	hipDeviceSynchronize();



///////////////////////////////////////////////////////////////////////////////////////////  
/////////////////////////Add ECORR Matrix/////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////// 	
/*

	if(incNGJitter > 0){

		int NGCoeffStartPoint = RF+DMF+incGroupNoise*GroupNoiseCoeff+incBandNoise*BandNoiseCoeff;

		add_EcorrToFMatrix<<< N, 1 >>>(FMatrix_d, GlobalEMatrix_d, N, NGCoeffStartPoint, numNGJitterEpochs);
	 	 
		hipDeviceSynchronize();

	}

*/
/*

///////////////////////////////////////////////////////////////////////////////////////////  
/////////////////////////Form the T Matrix/////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////// 	 


	//printf("entered 4\n");
	dimGrid.x=(T + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y; 

	if(D != NTime+NJumps){
		make_Tmatrix<<<N,1>>>(TMatrix_d, DMatrix_d, FMatrix_d, N, T, D, F);
	}
	else{

		make_Tmatrix<<<N,1>>>(TMatrix_d, GlobalStaticDmat_d, FMatrix_d, N, T, D, F);
	}

	hipDeviceSynchronize();
*/
///////////////////////////////////////////////////////////////////////////////////////////  
/////////////////////////Do the Algebra///////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////// 	
 	 
 	 
	//printf("entered 5 %i %i\n",T,N);
	
	
	dimGrid.x=(T + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (N + dimBlock.y - 1)/dimBlock.y;	

	MatMulKernel<<<dimGrid, dimBlock>>>(N,N,N, T,Noise_d,GlobalTotalMatrix_d,NT_d);
	//SimpleDiagMatMulKernel<<<1,1>>>(N, T, Noise_d, TMatrix_d, NT_d);
	hipDeviceSynchronize();

	double alpha=1.0;
	double beta=0.0; 
	

	status =  culaDeviceDgemm('T', 'N', T, T, N, alpha, GlobalTotalMatrix_d, N, NT_d, N, beta, TNT_d, T);
	checkStatus(status);

	hipDeviceSynchronize();
	

	status = culaDeviceDgemv('T', N, T, alpha, NT_d, N, resvec_d, 1, beta, NTd_d, 1);
	checkStatus(status);
	
	hipDeviceSynchronize();
	
	
	dimGrid.x=(T + dimBlock.x - 1)/dimBlock.x;
	dimGrid.y = (T + dimBlock.y - 1)/dimBlock.y;	
	

	addCoeffsKernel<<<1,1>>>(T,D,F,TNT_d,powercoeff_d);

	hipDeviceSynchronize();
	
	
	//printf("entered 6: %i \n", T);
	double *dettemp_d;
	double *tempval=new double[1];
	err = hipMalloc( (void **)&dettemp_d, sizeof(double) );
	checkCudaError(err);

	int carryOn=0;
	status = culaDeviceDpotrf('L', T, TNT_d, T);
	hipDeviceSynchronize();
	//printf("entered 6.25\n");
	
	checkStatusCarryOn(status,carryOn);
	
	hipDeviceSynchronize();

	//printf("entered 6.5\n");
	calc_det<<< 1, 1 >>>( TNT_d, dettemp_d, T);
	err = hipMemcpy( tempval, dettemp_d, sizeof(double), hipMemcpyDeviceToHost);
	checkCudaError(err);
	likevals[0]=tempval[0];
	
	hipDeviceSynchronize();
     
	if(carryOn == 1){

		hipFree(Freqs_d);
		hipFree(ObsFreqs_d);
		hipFree(powercoeff_d);
		hipFree(resvec_d);
		hipFree(BATvec_d);
		hipFree(Noise_d);
		hipFree(DMVec_d);
		hipFree(SysGroups_d);

//		hipFree(FMatrix_d);
		if(D != NTime+NJumps){hipFree(DMatrix_d);}
//		hipFree(TMatrix_d);
		hipFree(NT_d);
		hipFree(TNT_d);
		hipFree(NTd_d);
		hipFree(BandInfo_d);
		
		hipFree(dettemp_d);
		delete(tempval);
		return;
	}

	//printf("entered 7\n");
	double *WorkVec_d;
	err = hipMalloc( (void **)&WorkVec_d, sizeof(double)*T );
	checkCudaError(err);
	copyvec<<< T, 1 >>>(WorkVec_d, NTd_d, T);
	
	hipDeviceSynchronize();
	

	status=culaDeviceDpotrs('L', T, 1, TNT_d, T, WorkVec_d, T);
	checkStatus(status);
	
	hipDeviceSynchronize();
	
	double *freqlike_d;
	err = hipMalloc( (void **)&freqlike_d, sizeof(double));
	checkCudaError(err);

	calc_DotLike<<< 1, 1 >>>(WorkVec_d, NTd_d, T, freqlike_d);
	
	hipDeviceSynchronize();
	err = hipMemcpy( tempval, freqlike_d, sizeof(double), hipMemcpyDeviceToHost);
	checkCudaError(err);
	likevals[1]=tempval[0];

	
	//printf("entered 8\n");
	hipFree(Freqs_d);
	hipFree(ObsFreqs_d);
	hipFree(powercoeff_d);
	hipFree(resvec_d);
	hipFree(BATvec_d);
	hipFree(Noise_d);
	hipFree(DMVec_d);	
	hipFree(SysGroups_d);

//	hipFree(FMatrix_d);
	if(D != NTime+NJumps){hipFree(DMatrix_d);}
//	hipFree(TMatrix_d);
	hipFree(NT_d);
	hipFree(TNT_d);
	hipFree(NTd_d);
	hipFree(BandInfo_d);
	
	hipFree(WorkVec_d);
	hipFree(dettemp_d);
	hipFree(freqlike_d);
	delete(tempval);
	
	//printf("entered 9\n");
		
}
 	 


extern "C" void copy_staticTmat_(double *T, int totalsize, int Nobs){

    hipError_t err;

 	 err = hipMalloc( (void **)&GlobalTotalMatrix_d, sizeof(double)*totalsize*Nobs );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalTotalMatrix_d, T, sizeof(double)*totalsize*Nobs, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}


/*

 	 

 extern "C" void copy_floatgmat_(float *G, int N){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalGmatFloat_d, sizeof(float)*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalGmatFloat_d, G, sizeof(float)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}


extern "C" void copy_gmat_(double *G, int N){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalGmat_d, sizeof(double)*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalGmat_d, G, sizeof(double)*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}

extern "C" void copy_staticgmat_(double *G, int M, int N){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalStaticGmat_d, sizeof(double)*N*M );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalStaticGmat_d, G, sizeof(double)*N*M, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}

extern "C" void copy_staticumat_(double *G, int M, int N){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");


 	 err = hipMalloc( (void **)&GlobalStaticUGmat_d, sizeof(double)*M*N );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalStaticUGmat_d, G, sizeof(double)*M*N, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}

extern "C" void copy_staticdmat_(double **TNDM, double *TNDMVec, int N, int D){

    hipError_t err;

   // Allocate memory on GPU
	//printf("copying G\n");
	
	err = hipMalloc( (void **)&GlobalStaticDmat_d, sizeof(double)*N*D );
	checkCudaError(err);

	err = hipMemcpy(GlobalStaticDmat_d, TNDMVec, sizeof(double)*D*N, hipMemcpyHostToDevice );
	checkCudaError(err);
	
	double *U_d;
	double *V_d;
	double *S_d;
	
	err = hipMalloc( (void **)&U_d, sizeof(double)*N*N );
	checkCudaError(err);
	err = hipMalloc( (void **)&V_d, sizeof(double)*D*D );
	checkCudaError(err);
	err = hipMalloc( (void **)&S_d, sizeof(double)*D );
	checkCudaError(err);
	
	
	culaDeviceDgesvd('O','N', N, D, GlobalStaticDmat_d, N, S_d, U_d, N, V_d, D);
	
	hipFree(V_d);
	hipFree(S_d);
	hipFree(U_d);
	
    hipDeviceSynchronize();
    

}

extern "C" void copy_staticECorrmat_(double *E, int EcorrSize, int Nobs){

    hipError_t err;

 	 err = hipMalloc( (void **)&GlobalEMatrix_d, sizeof(double)*EcorrSize*Nobs );
	 checkCudaError(err);

     // copy vectors from CPU to GPU
 	 err = hipMemcpy( GlobalEMatrix_d, E, sizeof(double)*EcorrSize*Nobs, hipMemcpyHostToDevice );
 	 checkCudaError(err);

   return;
}

*/

